
#include <hip/hip_runtime.h>
// takes output grayscale mask from Otsu's method
// pass a filter (structural element) over the data and reduce the value of each pixel 
// based on interactions with the values of its neighbors

// for input matrix A and structural element matrix B:
// for each pixel in A, superimpose the origin of B. 
// if B is completely contained by A (i.e. every pixel of B = corresponding pixel of A),
// then the pixel is retained (1), else it is deleted (0).

// basic erosion using global memory
__global__ void image_erode(unsigned char* inImage, unsigned char* outImage_shadow, unsigned char* outImage_light, int mask_width, int width, int height) {
    
    int col = threadIdx.x + blockIdx.x * blockDim.x; // column (x-direction) index
    int row = threadIdx.y + blockIdx.y * blockDim.y; // row (y-direction) index

    if (col < width && row < height) {
        int startRow = row - (mask_width/2);
        int startCol = col - (mask_width/2);

        unsigned char value_shadow = 1;
        unsigned char value_light = 1;

        for (int j = 0; j < mask_width; j++) {      // row
            for (int k = 0; k < mask_width; k++) {  // column
                int curRow = startRow + j;
                int curCol = startCol + k;

                if((curRow >= 0 && curRow < height) && (curCol >= 0 && curCol < width)) { // check that pixel is in valid range
                    // output pixel value is the min value of all pixels in the neighborhood
                    // pixel is set to 0 if any of the neighboring pixels have the value 0
                    value_shadow = min(value_shadow, inImage[curRow * width + curCol]);
                    value_light = min(value_light, 1 - inImage[curRow * width + curCol]);
                }
            }
        }

        outImage_shadow[row * width + col] = value_shadow;
        outImage_light[row * width + col] = value_light;
    }
}

// tiled erosion using shared memory
__global__ void image_erode_shared(unsigned char* inImage, unsigned char* outImage_shadow, unsigned char* outImage_light, int mask_width, int width, int height) {
    
    // mask width = 3
    // block size = 6x6
    // tile width = blockDim - (mask width - 1)
    //            = 6 - (3 - 1) = 4

    __shared__ float tile[20][20];
    int tile_width = 4;

    // thread indices
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // output indices
    int col = threadIdx.x + blockIdx.x * tile_width;
    int row = threadIdx.y + blockIdx.y * tile_width;

    // input indices
    int startRow = row - (mask_width/2);
    int startCol = col - (mask_width/2);

    if ((startRow >= 0 && startRow < height) && (startCol >= 0 && startCol < width)) {
        tile[ty][tx] = float(inImage[startRow * width + startCol]);
    }
    else tile[ty][tx] = 0.0;
    __syncthreads();

    if (tx < tile_width && ty < tile_width) {
        float value_shadow = 1;
        float value_light = 1;

        for (int j = 0; j < mask_width; j++) {
            for (int k = 0; k < mask_width; k++) {
                // FIXME: check
                value_shadow = min(value_shadow, tile[ty + j][tx + k]);
                value_light = min(value_light, 1 - tile[ty + j][tx + k]);
            }
        }

        if (col < width && row < height) {
            outImage_shadow[row * width + col] = value_shadow;
            outImage_light[row * width + col] = value_light;
        }
    }
}

// naive shadow mask kernel
__global__ void image_erode_shadow(unsigned char* inImage, unsigned char* outImage, int mask_width, int width, int height) {
    
    int col = threadIdx.x + blockIdx.x * blockDim.x; // column (x-direction) index
    int row = threadIdx.y + blockIdx.y * blockDim.y; // row (y-direction) index

    if (col < width && row < height) {
        int startRow = row - (mask_width/2);
        int startCol = col - (mask_width/2);

        unsigned char value = 1;

        for (int j = 0; j < mask_width; j++) {      // row
            for (int k = 0; k < mask_width; k++) {  // column
                int curRow = startRow + j;
                int curCol = startCol + k;

                if((curRow >= 0 && curRow < height) && (curCol >= 0 && curCol < width)) { // check that pixel is in valid range
                    // output pixel value is the min value of all pixels in the neighborhood
                    // pixel is set to 0 if any of the neighboring pixels have the value 0
                    value = min(value, inImage[curRow * width + curCol]);
                }
            }
        }

        outImage[row * width + col] = value;
    }
}

// naive light mask kernel
__global__ void image_erode_light(unsigned char* inImage, unsigned char* outImage, int mask_width, int width, int height) {
    
    int col = threadIdx.x + blockIdx.x * blockDim.x; // column (x-direction) index
    int row = threadIdx.y + blockIdx.y * blockDim.y; // row (y-direction) index

    if (col < width && row < height) {
        int startRow = row - (mask_width/2);
        int startCol = col - (mask_width/2);

        unsigned char value = 1;

        for (int j = 0; j < mask_width; j++) {      // row
            for (int k = 0; k < mask_width; k++) {  // column
                int curRow = startRow + j;
                int curCol = startCol + k;

                if ((curRow >= 0 && curRow < height) && (curCol >= 0 && curCol < width)) { // check that pixel is in valid range
                    // output pixel value is the min value of all pixels in the neighborhood
                    // pixel is set to 0 if any of the neighboring pixels have the value 0
                    value = min(value, 1 - inImage[curRow * width + curCol]);
                }
            }
        }

        outImage[row * width + col] = value;
    }
}