#include "hip/hip_runtime.h"
// takes output grayscale mask from Otsu's method
// pass a filter (structural element) over the data and reduce the value of each pixel 
// based on interactions with the values of its neighbors

// for input matrix A and structural element matrix B:
// for each pixel in A, superimpose the origin of B. 
// if B is completely contained by A (i.e. every pixel of B = corresponding pixel of A),
// then the pixel is retained (1), else it is deleted (0).


__global__ void image_erode(unsigned char* inImage, float* outImage_light, float* outImage_shadow, int mask_width, int width, int height) {
    
    int col = threadIdx.x + blockIdx.x * blockDim.x; // column (x-direction) index
    int row = threadIdx.y + blockIdx.x * blockDim.y; // row (y-direction) index

    if (col < width && row < height) {
        float value = 1;
        int startRow = row - (mask_width/2);
        int startCol = col - (mask_width/2);

        for (int j = 0; j < mask_width; j++) {      // row
            for (int k = 0; k < mask_width; k++) {  // column
                int curRow = startRow + j;
                int curCol = startCol + k;

                if((curRow >= 0 && curRow < height) && (curCol >= 0 && curCol < width)) { // check that pixel is in valid range
                    value = min(value, inImage[i * width +j]); // FIXME: check operation
                }
            }
        }

        outImage_shadow[row * width + col] = value;
        // outImage_light[]
    }

    unsigned int start_i = max(y - radio, 0);
    unsigned int end_i = min(height - 1, y + radio);
    unsigned int start_j = max(x - radio, 0);
    unsigned int end_j = min(width - 1, x + radio);
    int value = 255;
    for (int i = start_i; i <= end_i; i++) {
        for (int j = start_j; j <= end_j; j++) {
            value = min(value, src[i * width + j]);
        }
    }
    dst[y * width + x] = value;

    // based on MATLAB imerode function
    // strel = [1 1 1 1 1; 1 1 1 1 1; 1 1 1 1 1; 1 1 1 1 1; 1 1 1 1 1];
    // eroded_gray_shadow_mask = imerode(gray_mask, strel);
    // eroded_gray_light_mask = imerode(1-gray_mask, strel);
}