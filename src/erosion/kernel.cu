
#include <hip/hip_runtime.h>
// takes output grayscale mask from Otsu's method
// pass a filter (structural element) over the data and reduce the value of each pixel 
// based on interactions with the values of its neighbors

// for input matrix A and structural element matrix B:
// for each pixel in A, superimpose the origin of B. 
// if B is completely contained by A (i.e. every pixel of B = corresponding pixel of A),
// then the pixel is retained (1), else it is deleted (0).


__global__ void image_erode(unsigned char* inImage, unsigned char* outImage_shadow, unsigned char* outImage_light, int mask_width, int width, int height) {
    
    int col = threadIdx.x + blockIdx.x * blockDim.x; // column (x-direction) index
    int row = threadIdx.y + blockIdx.y * blockDim.y; // row (y-direction) index

    if (col < width && row < height) {
        int startRow = row - (mask_width/2);
        int startCol = col - (mask_width/2);

        unsigned char value_shadow = 1;
        unsigned char value_light = 1;

        for (int j = 0; j < mask_width; j++) {      // row
            for (int k = 0; k < mask_width; k++) {  // column
                int curRow = startRow + j;
                int curCol = startCol + k;

                if((curRow >= 0 && curRow < height) && (curCol >= 0 && curCol < width)) { // check that pixel is in valid range
                    // output pixel value is the min value of all pixels in the neighborhood
                    // pixel is set to 0 if any of the neighboring pixels have the value 0
                    value_shadow = min(value_shadow, inImage[curRow * width + curCol]);
                    value_light = min(value_light, 1 - inImage[curRow * width + curCol]);
                }
            }
        }

        outImage_shadow[row * width + col] = value_shadow;
        outImage_light[row * width + col] = value_light;
    }

    // based on MATLAB imerode function
    // strel = [1 1 1 1 1; 1 1 1 1 1; 1 1 1 1 1; 1 1 1 1 1; 1 1 1 1 1];
    // eroded_gray_shadow_mask = imerode(gray_mask, strel);
    // eroded_gray_light_mask = imerode(1-gray_mask, strel);
}