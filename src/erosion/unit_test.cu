#include <wb.h>
#include "kernel.cu"
#include "../globals.h"

int main(void) {

  int imageWidth;
  int imageHeight;
  int imageSize;

  unsigned char *hostInputImage;
  float *hostOutputImage_shadow;
  float *hostOutputImage_light;

  unsigned char *deviceInputImage;
  float *deviceOutputImage_shadow;
  float *deviceOutputImage_light;

  // float *hostMask;
  // float *deviceMask;

  // int maskWidth = 1;
  // int imageHeight = 2;
  // int imageWidth = 2;

  imageSize = imageHeight * imageWidth;

  hostInputImage = (unsigned char *)malloc(imageSize * sizeof(unsigned char));
  hostOutputImage_shadow = (float *)malloc(imageSize * sizeof(float));
  hostOutputImage_light = (float *)malloc(imageSize * sizeof(float));
  // hostMask = (float *)malloc(maskWidth * maskWidth * sizeof(float));

  printf("\n\ninput image:\t");
  for(int i = 0; i < imageSize; i++){
    hostInputImage[i] = 1;
    printf("%d, ", hostInputImage[i]); 
  }
  printf("\n\n");
  
  printf("\n\nmask:\t");
  for(int i = 0; i < maskWidth*maskWidth; i++){
    hostMask[i] = 0.5;
    printf("%f, ", hostMask[i]); 
  }
  printf("\n\n");

  wbTime_start(GPU, "Allocating GPU memory.");
  CUDA_CHECK( hipMalloc((void **)&deviceInputImage, imageSize * sizeof(unsigned char)) );
  CUDA_CHECK( hipMalloc((void **)&deviceOutputImage_shadow, imageSize * sizeof(float)) );
  CUDA_CHECK( hipMalloc((void **)&deviceOutputImage_light, imageSize * sizeof(float)) );
  // CUDA_CHECK( hipMalloc((void **)&deviceMask, maskWidth * maskWidth * sizeof(float)) );
  CUDA_CHECK( hipDeviceSynchronize() );
  wbTime_stop(GPU, "Allocating GPU memory.");

  //@@ Copy memory to the GPU here
  wbTime_start(GPU, "Copying input memory to the GPU.");
  CUDA_CHECK(hipMemcpy(deviceInputImage, hostInputImage, imageSize * sizeof(unsigned char),
                        hipMemcpyHostToDevice));
  // CUDA_CHECK(hipMemcpy(deviceMask, hostMask, maskWidth * maskWidth * sizeof(float),
  //                       hipMemcpyHostToDevice));                        
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  dim3 blockDim1(8,8), gridDim1(1,1);
  conv2d<<<gridDim1, blockDim1>>>(deviceInputImage, deviceOutputImage_shadow, 
                                  deviceOutputImage_light, maskWidth, imageWidth, 
                                  imageHeight);

  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  //@@ Copy the GPU memory back to the CPU here
  wbTime_start(Copy, "Copying output memory to the CPU");
  CUDA_CHECK(hipMemcpy(hostOutputImage_shadow, deviceOutputImage_shadow,
                        imageSize * sizeof(float),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(hostOutputImage_light, deviceOutputImage_light,
                        imageSize * sizeof(float),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(Copy, "Copying output memory to the CPU");

  printf("\n\noutput image (shadow):\t");
  for(int i = 0; i < imageSize; i++){
      printf("%.5f, ", hostOutputImage_shadow[i]);
  }
  printf("\n\n");

  printf("\n\noutput image (light):\t");
  for(int i = 0; i < imageSize; i++){
      printf("%.5f, ", hostOutputImage_light[i]);
  }
  printf("\n\n");  

  CUDA_CHECK(hipFree(deviceInputImage));
  CUDA_CHECK(hipFree(deviceOutputImage_shadow));
  CUDA_CHECK(hipFree(deviceOutputImage_light));

  return 0;
}