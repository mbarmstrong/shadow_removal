#define SOLUTION

#include <wb.h>
#include "globals.h"
#include "./color_conversion/launch.cu"
#include "./otsu_method/launch.cu"
#include "./erosion/launch.cu"
#include "./convolution/launch.cu"
#include "./result_integration/launch.cu"


void execute_shadow_removal(float *rgbImage, int imageWidth, int imageHeight, char* outDir){


  int imageSize = imageWidth * imageHeight;

  // setup end to end timer
  hipEvent_t astartEvent, astopEvent;
  float aelapsedTime;
  hipEventCreate(&astartEvent);
  hipEventCreate(&astopEvent);

  hipEventRecord(astartEvent, 0);

  //--------------------------------------------------
  // execute color conversion 
  // generate three images: color invarient, gray and YUV
  //--------------------------------------------------
  float *invImage;
  unsigned char *grayImage;
  unsigned char *yuvImage;

  invImage =  (float *)malloc(imageSize * NUM_CHANNELS * sizeof(float));
  grayImage = (unsigned char *)malloc(imageSize * 1 * sizeof(unsigned char));
  yuvImage =  (unsigned char *)malloc(imageSize * NUM_CHANNELS * sizeof(unsigned char));

  // execute color convert to get grey and yuv images, note this transposes the output yuv image in memory
  // so all channels store their pixels sequentially, for example all the y pixels followed by all the
  // u pixels then folled by all the v pixels for the yuv image
  launch_color_convert(rgbImage, invImage, grayImage, yuvImage, imageWidth, imageHeight, imageSize, "convert");

  //--------------------------------------------------
  // execute otsu's method
  // using U channel of YUV and grayscale image
  //--------------------------------------------------
  unsigned char *grayMask;
  unsigned char *yuvMask;
  unsigned char *u = yuvImage + 1*imageSize; //get second channel yuv image
  float level_gray = 0.0;
  float level_u = 0.0;

  // allocate host memory for gray and yuv masks
  grayMask = (unsigned char *)malloc(imageSize * sizeof(unsigned char));
  yuvMask = (unsigned char *)malloc(imageSize * sizeof(unsigned char));

  // calculate gray threshold and binarize to get the gray mask using gpu kernels
  level_gray = launch_otsu_method(grayImage, imageWidth, imageHeight, "gray");
  launch_image_binarization(grayImage, grayMask, level_gray, imageWidth, imageHeight, true, "gray");

  // calculate u threshold and binarize to get the yuv mask using gpu kernels
  level_u = launch_otsu_method(u, imageWidth, imageHeight, "yuv");
  launch_image_binarization(u, yuvMask, level_u, imageWidth, imageHeight, false, "yuv");


  //--------------------------------------------------
  // execute erosion using gray mask
  //
  //--------------------------------------------------
  unsigned char *erodedShadow;
  unsigned char *erodedLight;
  int maskWidth = 5;

  // allocate host memory for eroded shadow mask and eroded light mask
  erodedShadow = (unsigned char *)malloc(imageSize * sizeof(unsigned char));
  erodedLight = (unsigned char *)malloc(imageSize * sizeof(unsigned char));

  // launch erosion kernels
  launch_erosion(grayMask, erodedShadow, erodedLight, maskWidth, imageWidth, imageHeight);


  //--------------------------------------------------
  // execute convolution using yuv mask
  //
  //--------------------------------------------------
  float *smoothMask;

  // allocate host memory for smooth mask
  smoothMask = (float *)malloc(imageSize * sizeof(float));

  // launch convolution kernels
  launch_convolution(yuvMask, smoothMask, maskWidth, imageWidth, imageHeight);


  //--------------------------------------------------
  //  Execute Result Integration method -
  //  using original image, gray shadow,gray Light,
  //  Eroded shadow, eroded light and smooth mask
  //--------------------------------------------------
  float *finalImage;

  // allocate memory for final image
  finalImage = (float *)malloc(imageSize * NUM_CHANNELS * sizeof(float));

  launch_result_integration(rgbImage,erodedShadow,erodedLight,smoothMask,finalImage,imageWidth,imageHeight);

  hipEventRecord(astopEvent, 0);
  hipEventSynchronize(astopEvent);
  hipEventElapsedTime(&aelapsedTime, astartEvent, astopEvent);

  // debug prints for verifying each step
  #if PRINT_DEBUG

    int debugPixelRow = 0;
    int debugPixelCol = 0;

    printf("\nInital RGB Image:");
    print_pixel(rgbImage,debugPixelRow,debugPixelCol,1,3,imageSize);

    printf("\nGray Image:");
    print_pixel(grayImage,debugPixelRow,debugPixelCol,1,1,imageSize);

    printf("\nU Image:");
    print_pixel(u,debugPixelRow,debugPixelCol,1,1,imageSize);

    printf("\n\nGray Level:\t%.4f\n\n",level_gray);
    printf("\n\nYUV Level:\t%.4f\n\n",level_u);

    printf("\nGray Mask:");
    print_pixel(grayMask,debugPixelRow,debugPixelCol,1,1,imageSize);

    printf("\nYUV Mask:");
    print_pixel(yuvMask,debugPixelRow,debugPixelCol,1,1,imageSize);

    printf("\nShadow Mask:");
    print_pixel(erodedShadow,debugPixelRow,debugPixelCol,1,1,imageSize);
  
    printf("\nLight Mask:");
    print_pixel(erodedLight,debugPixelRow,debugPixelCol,1,1,imageSize);
  
    printf("\nSmooth Mask:");
    print_pixel(smoothMask,debugPixelRow,debugPixelCol,1,1,imageSize);

    printf("\nFinal Image:");
    print_pixel(finalImage,debugPixelRow,debugPixelCol,1,3,imageWidth);


    char *output_file_name;

    // write images for each step of the shadow removal process
    output_file_name = wbPath_join(outDir, "input.ppm");
    write_image(output_file_name,rgbImage,imageWidth,imageHeight,NUM_CHANNELS);

    output_file_name = wbPath_join(outDir, "greyImage.ppm");
    write_image(output_file_name,grayImage,imageWidth,imageHeight,false);

    output_file_name = wbPath_join(outDir, "U.ppm");
    write_image(output_file_name,u,imageWidth,imageHeight,false);

    output_file_name = wbPath_join(outDir, "grayMask.ppm");
    write_image(output_file_name,grayMask,imageWidth,imageHeight,true);

    output_file_name = wbPath_join(outDir, "yuvMask.ppm");
    write_image(output_file_name,yuvMask,imageWidth,imageHeight,true);

    output_file_name = wbPath_join(outDir, "erodedShadow.ppm");
    write_image(output_file_name,erodedShadow,imageWidth,imageHeight,true);

    output_file_name = wbPath_join(outDir, "erodedLight.ppm");
    write_image(output_file_name,erodedLight,imageWidth,imageHeight,true);

    output_file_name = wbPath_join(outDir, "smoothMask.ppm");
    write_image(output_file_name,smoothMask,imageWidth,imageHeight,1);

    output_file_name = wbPath_join(outDir, "output.ppm");
    write_image(output_file_name,finalImage,imageWidth,imageHeight,NUM_CHANNELS);

  #endif

  printf("Done! Total Execution Time (ms):\t%f\n\n",aelapsedTime);
 
  // cleanup host mem
  free(invImage);
  free(grayImage);
  free(yuvImage);
  free(grayMask);
  free(yuvMask);
  free(erodedShadow);
  free(erodedLight);
  free(smoothMask);
  free(finalImage);

}

int main(int argc, char *argv[]) {

  //-------------------------------------------------
  //  get inputs and load inital rgb image
  //
  //-------------------------------------------------
  wbArg_t args;

  char *inputImageFile;
  //char *outputImageFile;

	wbImage_t inputImage_RGB;
  float *rgbImage;
  int imageWidth;
  int imageHeight;

  args = wbArg_read(argc, argv); // parse the input arguments

  char *outputDir = wbArg_getOutputFile(args);
  char *outputFile = wbPath_join(outputDir, "kernel_times.csv");

  timerLog = timerLog_new(outputFile); //setup global instance of logger

  int inputFileCount = wbArg_getInputCount(args);

  // loop through all the input files and run the shadow removal algorithm 
  for(int i = 0; i < inputFileCount; i++) {

    // read image
    inputImageFile = wbArg_getInputFile(args, i);
    inputImage_RGB = wbImport(inputImageFile);

    // load image from inputs and get data
    imageWidth = wbImage_getWidth(inputImage_RGB);
    imageHeight = wbImage_getHeight(inputImage_RGB);

    rgbImage = wbImage_getData(inputImage_RGB);

    printf("\nRunning shadow removal on image of %dx%d... ",
          imageWidth, imageHeight, NUM_CHANNELS);

    // call shadow removal on inital rgb image
    execute_shadow_removal(rgbImage, imageWidth, imageHeight, outputDir);
  }

  timerLog_save(&timerLog); //save kernel times to output file

  wbImage_delete(inputImage_RGB);
  
  return 0;
}
