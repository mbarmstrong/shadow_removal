#include "hip/hip_runtime.h"
#include <wb.h>
#include "kernel.cu"
#include "kernel_reduction.cu"
#include "../globals.h"


void launch_result_integration(float *rgbImage,unsigned char *erodedShadowMask,unsigned char *erodedLightMask, 
  float *smoothMask,float *finalImage,int imageWidth, int imageHeight) {
  
    float redSumShadowArray;
    float greenSumShadowArray;
    float blueSumShadowArray;
    float redSumLightArray; 
    float greenSumLightArray;
    float blueSumLightArray;
    float erodedSumShadowArray;
    float erodedSumLightArray;
    float *deviceRgbImage;
    float *deviceRedShadowArray;
    float *deviceGreenShadowArray;
    float *deviceBlueShadowArray;
    float *deviceRedLightArray;
    float *deviceGreenLightArray;
    float *deviceBlueLightArray;
    unsigned char *deviceErodedShadowMask;
    unsigned char *deviceErodedLightMask;
    float *deviceRedRatio;
    float *deviceGreenRatio;
    float *deviceBlueRatio;
    float *deviceSmoothMask;
    float *deviceFinalImage;
  
    int imageSize = imageHeight * imageWidth;
    int n_threads = 16;

    hipEvent_t astartEvent1, astopEvent1;
    float aelapsedTime1;
    hipEventCreate(&astartEvent1);
    hipEventCreate(&astopEvent1);

  
    wbTime_start(GPU, "Allocating GPU memory.");
    CUDA_CHECK(hipMalloc((void **)&deviceRgbImage, imageSize * NUM_CHANNELS * sizeof(float)));
    CUDA_CHECK(hipMalloc((void **)&deviceSmoothMask, imageSize * sizeof(float)));
    CUDA_CHECK(hipMalloc((void **)&deviceErodedShadowMask, imageSize * sizeof(unsigned char)));
    CUDA_CHECK(hipMalloc((void **)&deviceErodedLightMask, imageSize * sizeof(unsigned char)));
    CUDA_CHECK(hipMalloc((void **)&deviceRedShadowArray, imageSize * sizeof(float)));
    CUDA_CHECK(hipMalloc((void **)&deviceGreenShadowArray, imageSize * sizeof(float)));
    CUDA_CHECK(hipMalloc((void **)&deviceBlueShadowArray, imageSize * sizeof(float)));
    CUDA_CHECK(hipMalloc((void **)&deviceRedLightArray, imageSize * sizeof(float)));
    CUDA_CHECK(hipMalloc((void **)&deviceGreenLightArray, imageSize * sizeof(float)));
    CUDA_CHECK(hipMalloc((void **)&deviceBlueLightArray, imageSize * sizeof(float)));     
    CUDA_CHECK(hipDeviceSynchronize());
    wbTime_stop(GPU, "Allocating GPU memory.");
  
    // Copy memory to the GPU here
    wbTime_start(GPU, "Copying input memory to the GPU.");
    CUDA_CHECK(hipMemcpy(deviceRgbImage, rgbImage, imageSize * NUM_CHANNELS * sizeof(float),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(deviceSmoothMask, smoothMask, imageSize * sizeof(float),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(deviceErodedShadowMask, erodedShadowMask, imageSize * sizeof(unsigned char),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(deviceErodedLightMask, erodedLightMask, imageSize * sizeof(unsigned char),
                          hipMemcpyHostToDevice));    
    CUDA_CHECK(hipDeviceSynchronize());
    wbTime_stop(GPU, "Copying input memory to the GPU.");
  
  // Launch multiple_rgbImage_byMask kernel on the bins
  
    dim3 gridDim(ceil((float)imageWidth/(float)n_threads),ceil((float)imageHeight/(float)n_threads),1);
    dim3 blockDim(n_threads,n_threads,1);
    hipEventRecord(astartEvent1, 0);
    multiply_rgbImage_byMask<<<gridDim, blockDim>>>(
      deviceRgbImage, deviceErodedShadowMask, 
      deviceErodedLightMask, deviceRedShadowArray,deviceGreenShadowArray,deviceBlueShadowArray,
      deviceRedLightArray,deviceGreenLightArray,deviceBlueLightArray,imageWidth,imageHeight, NUM_CHANNELS);
    hipEventRecord(astopEvent1, 0);
    hipEventSynchronize(astopEvent1);
    hipEventElapsedTime(&aelapsedTime1, astartEvent1, astopEvent1);  

    printf("\nDone! Total Execution Time for Result Integration Kernel1 (ms):\t%f\n\n",aelapsedTime1);

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

 // Launch sum_up_arrays kernel on the light and shadow arrays for each channelth
  hipEventRecord(astartEvent1, 0);
  redSumShadowArray = gpu_sum_reduce(deviceRedShadowArray, imageSize);

  hipEventRecord(astopEvent1, 0);
  hipEventSynchronize(astopEvent1);
  hipEventElapsedTime(&aelapsedTime1, astartEvent1, astopEvent1);  

  printf("\nDone! Total Execution Time for Result Integration Kernel2 (ms):\t%f\n\n",aelapsedTime1);

  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  greenSumShadowArray = gpu_sum_reduce(deviceGreenShadowArray, imageSize);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

 // Launch sum_up_arrays kernel on the light and shadow arrays for each channel
  blueSumShadowArray = gpu_sum_reduce(deviceBlueShadowArray, imageSize);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());    

 // Launch sum_up_arrays kernel on the shadow arrays for each channel
  redSumLightArray = gpu_sum_reduce(deviceRedLightArray, imageSize);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());    

 // Launch sum_up_arrays kernel on the light arrays for each channel
  greenSumLightArray = gpu_sum_reduce(deviceGreenLightArray, imageSize);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());    

  blueSumLightArray = gpu_sum_reduce(deviceBlueLightArray, imageSize); 
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());      

// Launch sum_up_arrays kernel on the eroded shadow array
  erodedSumShadowArray = gpu_sum_reduce(deviceErodedShadowMask, imageSize);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

// Launch sum_up_arrays kernel on the eroded light array
erodedSumLightArray = gpu_sum_reduce(deviceErodedLightMask, imageSize); 
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());  

if(PRINT_DEBUG){
  printf("\nSum of Red Shadow Array:\t %.04f",redSumShadowArray);
  printf("\nSum of Green Shadow Array:\t%.04f",greenSumShadowArray);
  printf("\nSum of Blue Shadow Array:\t %.04f",blueSumShadowArray);
  printf("\nSum of Red Light Array:\t %.04f",redSumLightArray);
  printf("\nSum of Green Light Array:\t %.04f",greenSumLightArray);
  printf("\nSum of Blue Light Array:\t%.04f",blueSumShadowArray);
  printf("\nSum of Eroded  Shadow Array:\t%.04f",erodedSumShadowArray);
  printf("\nSum of Eroded  Light Array:\t%.04f",erodedSumLightArray);
  printf("\n");
}

  float redRatio = (((redSumLightArray/erodedSumLightArray)/(redSumShadowArray/erodedSumShadowArray)) -1);
  float greenRatio = (((greenSumLightArray/erodedSumLightArray)/(greenSumShadowArray/erodedSumShadowArray)) -1);
  float blueRatio = (((blueSumLightArray/erodedSumLightArray)/(blueSumShadowArray/erodedSumShadowArray)) -1);
  
  wbTime_start(GPU, "Allocating GPU memory.");
  CUDA_CHECK( hipMalloc((void **)&deviceFinalImage, imageSize * NUM_CHANNELS * sizeof(float))); 
      CUDA_CHECK(hipGetLastError());
  CUDA_CHECK( hipMalloc((void **)&deviceSmoothMask, imageSize * sizeof(float)));      
      CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());
      CUDA_CHECK(hipGetLastError());
  wbTime_stop(GPU, "Allocating GPU memory.");


  wbTime_start(GPU, "Allocating GPU memory.");
  CUDA_CHECK( hipMalloc((void **)&deviceRedRatio, sizeof(float)));   
  CUDA_CHECK( hipMalloc((void **)&deviceGreenRatio, sizeof(float)));   
  CUDA_CHECK( hipMalloc((void **)&deviceBlueRatio, sizeof(float)));    
  CUDA_CHECK(hipDeviceSynchronize());
  CUDA_CHECK(hipGetLastError()); 
  wbTime_stop(GPU, "Allocating GPU memory."); 

  // Copy the GPU memory back to the CPU here
wbTime_start(Copy, "Copying host memory to the GPU");
CUDA_CHECK(hipMemcpy(deviceRedRatio, &redRatio,
                      sizeof(float),
                      hipMemcpyHostToDevice));
CUDA_CHECK(hipMemcpy(deviceGreenRatio, &greenRatio,
                      sizeof(float),
                      hipMemcpyHostToDevice));
CUDA_CHECK(hipMemcpy(deviceBlueRatio, &blueRatio,
                      sizeof(float),
                      hipMemcpyHostToDevice));              
CUDA_CHECK(hipDeviceSynchronize());
wbTime_stop(Copy, "Copying output memory to the CPU");
  
  // zero out bins
  CUDA_CHECK(hipMemset(deviceFinalImage, 0.0, imageSize * NUM_CHANNELS * sizeof(float)));

  // Launch calculate_rgb_ratio kernel on the eroded shadow array and calculates the final image
  dim3 gridDim2(ceil(imageWidth/16.0), ceil(imageHeight/16.0), 1);
  dim3 blockDim2(16, 16, 1);
  hipEventRecord(astartEvent1, 0);
  calculate_final_image_stride<<<gridDim2, blockDim2>>>(deviceRedRatio, deviceGreenRatio,deviceBlueRatio,
  deviceRgbImage, deviceSmoothMask, deviceFinalImage,
  imageWidth, imageHeight, NUM_CHANNELS);
   
  hipEventRecord(astopEvent1, 0);
  hipEventSynchronize(astopEvent1);
  hipEventElapsedTime(&aelapsedTime1, astartEvent1, astopEvent1);  

  printf("\nDone! Total Execution Time for Result Integration Kernel3 (ms):\t%f\n\n",aelapsedTime1);

  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  //@@ Copy the GPU memory back to the CPU here
  wbTime_start(Copy, "Copying output memory to the CPU");
  CUDA_CHECK(hipMemcpy(finalImage, deviceFinalImage,
                        imageSize * NUM_CHANNELS * sizeof(float),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(Copy, "Copying output memory to the CPU");
  
  // //@@ Free the GPU memory here
  wbTime_start(Copy, "Freeing GPU Memory");
  CUDA_CHECK(hipFree(deviceRgbImage));
  CUDA_CHECK(hipFree(deviceRedShadowArray));
  CUDA_CHECK(hipFree(deviceGreenShadowArray));
  CUDA_CHECK(hipFree(deviceBlueShadowArray));
  CUDA_CHECK(hipFree(deviceRedLightArray));
  CUDA_CHECK(hipFree(deviceGreenLightArray));
  CUDA_CHECK(hipFree(deviceBlueLightArray));
  CUDA_CHECK(hipFree(deviceErodedShadowMask));
  CUDA_CHECK(hipFree(deviceErodedLightMask));
  CUDA_CHECK(hipFree(deviceRedRatio));
  CUDA_CHECK(hipFree(deviceGreenRatio));
  CUDA_CHECK(hipFree(deviceBlueRatio));
  CUDA_CHECK(hipFree(deviceSmoothMask));
  CUDA_CHECK(hipFree(deviceFinalImage));
  wbTime_stop(Copy, "Freeing GPU Memory");

}

