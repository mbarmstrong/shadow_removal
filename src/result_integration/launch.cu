#include "hip/hip_runtime.h"
#include <wb.h>
#include "kernel.cu"
#include "../globals.h"


void launch_result_integration(float *rgbImage,unsigned char *erodedShadowMask,unsigned char *erodedLightMask, 
  float *smoothMask,float *finalImage,int imageWidth, int imageHeight) {
  
    float *redShadowArray;
    float *greenShadowArray;
    float *blueShadowArray;
    float *redLightArray;
    float *greenLightArray;
    float *blueLightArray;
    float *redSumShadowArray;
    float *greenSumShadowArray;
    float *blueSumShadowArray;
    float *redSumLightArray; 
    float *greenSumLightArray;
    float *blueSumLightArray;
    float *erodedSumShadowArray;
    float *erodedSumLightArray;
    float *deviceRgbImage;
    float *deviceRedShadowArray;
    float *deviceGreenShadowArray;
    float *deviceBlueShadowArray;
    float *deviceRedLightArray;
    float *deviceGreenLightArray;
    float *deviceBlueLightArray;
    unsigned char *deviceErodedShadowMask;
    unsigned char *deviceErodedLightMask;
    float *deviceRedSumShadowArray;
    float *deviceGreenSumShadowArray;
    float *deviceBlueSumShadowArray;
    float *deviceRedSumLightArray; 
    float *deviceGreenSumLightArray;
    float *deviceBlueSumLightArray;
    float *deviceErodedSumShadowArray;
    float *deviceErodedSumLightArray;
    float *deviceSmoothMask;
    float *deviceFinalImage;
  
    int imageSize = imageHeight * imageWidth;
    int n_threads = 16;
  
    wbTime_start(GPU, "Allocating GPU memory.");
    CUDA_CHECK(hipMalloc((void **)&deviceRgbImage, imageSize * sizeof(float)));
    CUDA_CHECK(hipMalloc((void **)&deviceSmoothMask, imageSize * sizeof(float)));
    CUDA_CHECK(hipMalloc((void **)&deviceErodedShadowMask, imageSize * sizeof(unsigned char)));
    CUDA_CHECK(hipMalloc((void **)&deviceErodedLightMask, imageSize * sizeof(unsigned char)));
    CUDA_CHECK(hipMalloc((void **)&deviceRedShadowArray, imageSize * sizeof(float)));
    CUDA_CHECK(hipMalloc((void **)&deviceGreenShadowArray, imageSize * sizeof(float)));
    CUDA_CHECK(hipMalloc((void **)&deviceBlueShadowArray, imageSize * sizeof(float)));
    CUDA_CHECK(hipMalloc((void **)&deviceRedLightArray, imageSize * sizeof(float)));
    CUDA_CHECK(hipMalloc((void **)&deviceGreenLightArray, imageSize * sizeof(float)));
    CUDA_CHECK(hipMalloc((void **)&deviceBlueLightArray, imageSize * sizeof(float)));     
    CUDA_CHECK(hipDeviceSynchronize());
    wbTime_stop(GPU, "Allocating GPU memory.");
  
    // Copy memory to the GPU here
    wbTime_start(GPU, "Copying input memory to the GPU.");
    CUDA_CHECK(hipMemcpy(deviceRgbImage, rgbImage, imageSize * sizeof(float),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(deviceSmoothMask, smoothMask, imageSize * sizeof(float),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(deviceErodedShadowMask, erodedShadowMask, imageSize * sizeof(unsigned char),
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(deviceErodedLightMask, erodedLightMask, imageSize * sizeof(unsigned char),
                          hipMemcpyHostToDevice));    
    CUDA_CHECK(hipDeviceSynchronize());
    wbTime_stop(GPU, "Copying input memory to the GPU.");
  
  // Launch multiple_rgbImage_byMask kernel on the bins
  
    dim3 gridDim(ceil((float)imageWidth/(float)n_threads),ceil((float)imageHeight/(float)n_threads),1);
    dim3 blockDim(n_threads,n_threads,1);
    multiply_rgbImage_byMask<<<gridDim, blockDim>>>(
      deviceRgbImage, deviceErodedShadowMask, 
      deviceErodedLightMask, deviceRedShadowArray,deviceGreenShadowArray,deviceBlueShadowArray,
      deviceRedLightArray,deviceGreenLightArray,deviceBlueLightArray,imageWidth,imageHeight, NUM_CHANNELS);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
  

    redShadowArray = (float *)malloc(imageSize * sizeof(float));
    blueShadowArray = (float *)malloc(imageSize * sizeof(float));
    greenShadowArray = (float *)malloc(imageSize * sizeof(float));
    redLightArray = (float *)malloc(imageSize * sizeof(float));
    greenLightArray = (float *)malloc(imageSize * sizeof(float));
    blueLightArray = (float *)malloc(imageSize * sizeof(float));
   // Copy the GPU memory back to the CPU here
    wbTime_start(Copy, "Copying output memory to the CPU");
    CUDA_CHECK(hipMemcpy(redShadowArray, deviceRedShadowArray,
                          imageSize * sizeof(float),
                          hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(greenShadowArray, deviceGreenShadowArray,
                          imageSize * sizeof(float),
                          hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(blueShadowArray, deviceBlueShadowArray,
                          imageSize * sizeof(float),
                          hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(redLightArray, deviceRedLightArray,
                          imageSize * sizeof(float),
                          hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(greenLightArray, deviceGreenLightArray,
                          imageSize * sizeof(float),
                          hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(blueLightArray, deviceBlueLightArray,
                          imageSize * sizeof(float),
                          hipMemcpyDeviceToHost));                       
    CUDA_CHECK(hipDeviceSynchronize());
    wbTime_stop(Copy, "Copying output memory to the CPU");


    wbTime_start(GPU, "Allocating GPU memory.");
    CUDA_CHECK( hipMalloc((void **)&deviceRedSumShadowArray, imageSize * sizeof(float)));   
    CUDA_CHECK( hipMalloc((void **)&deviceGreenSumShadowArray, imageSize * sizeof(float)));   
    CUDA_CHECK( hipMalloc((void **)&deviceBlueSumShadowArray, imageSize * sizeof(float)));   
    CUDA_CHECK( hipMalloc((void **)&deviceRedSumLightArray, imageSize * sizeof(float)));   
    CUDA_CHECK( hipMalloc((void **)&deviceGreenSumLightArray, imageSize * sizeof(float)));
    CUDA_CHECK( hipMalloc((void **)&deviceBlueSumLightArray, imageSize * sizeof(float))); 
    CUDA_CHECK( hipMalloc((void **)&deviceErodedSumLightArray, imageSize * sizeof(float))); 
    CUDA_CHECK( hipMalloc((void **)&deviceErodedSumShadowArray, imageSize * sizeof(float)));         
    CUDA_CHECK(hipDeviceSynchronize());
    wbTime_stop(GPU, "Allocating GPU memory.");

 // Launch sum_up_arrays kernel on the light and shadow arrays for each channel
  dim3 gridDim2(ceil((float)imageWidth/(float)n_threads),ceil((float)imageHeight/(float)n_threads),1);
  dim3 blockDim2(16,16,1);
  sum_up_arrays_by_reduction<<<gridDim2, blockDim2, 256 * sizeof(float)>>>(
      deviceRedShadowArray,deviceRedSumShadowArray,imageSize);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

 // Launch sum_up_arrays kernel on the light and shadow arrays for each channel
  sum_up_arrays_by_reduction<<<gridDim2, blockDim2, 256 * sizeof(float)>>>(
      deviceGreenShadowArray,deviceGreenSumShadowArray,imageSize);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());    

 // Launch sum_up_arrays kernel on the shadow arrays for each channel
  sum_up_arrays_by_reduction<<<gridDim2, blockDim2, 256 * sizeof(float)>>>(
      deviceBlueShadowArray,deviceBlueSumShadowArray,imageSize);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());    

 // Launch sum_up_arrays kernel on the light arrays for each channel
  sum_up_arrays_by_reduction<<<gridDim2, blockDim2, 256 * sizeof(float)>>>(
      deviceRedLightArray,deviceRedSumLightArray,imageSize);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());    

  sum_up_arrays_by_reduction<<<gridDim2, blockDim2, 256 * sizeof(float)>>>(
      deviceGreenLightArray,deviceGreenSumLightArray,imageSize);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());    

  sum_up_arrays_by_reduction<<<gridDim2, blockDim2, 256 * sizeof(float)>>>(
      deviceBlueLightArray,deviceBlueSumLightArray,imageSize);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());    

// Launch sum_up_arrays kernel on the eroded shadow array
  sum_up_arrays_by_reduction1<<<gridDim2, blockDim2, 256 * sizeof(float)>>>(
      deviceErodedShadowMask,deviceErodedSumShadowArray,imageSize);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

// Launch sum_up_arrays kernel on the eroded light array
  sum_up_arrays_by_reduction1<<<gridDim2, blockDim2, 256 * sizeof(float)>>>(
    deviceErodedLightMask,deviceErodedSumLightArray,imageSize);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());  

redSumShadowArray = (float *)malloc(imageSize * sizeof(float));
greenSumShadowArray = (float *)malloc(imageSize * sizeof(float));
blueSumShadowArray = (float *)malloc(imageSize * sizeof(float));
redSumLightArray = (float *)malloc(imageSize * sizeof(float));
greenSumLightArray = (float *)malloc(imageSize * sizeof(float));
blueSumLightArray = (float *)malloc(imageSize * sizeof(float));
erodedSumShadowArray = (float *)malloc(imageSize * sizeof(float));
erodedSumLightArray = (float *)malloc(imageSize * sizeof(float));

// Copy the GPU memory back to the CPU here
wbTime_start(Copy, "Copying output memory to the CPU");
CUDA_CHECK(hipMemcpy(redSumShadowArray, deviceRedSumShadowArray,
                      imageSize * sizeof(float),
                      hipMemcpyDeviceToHost));
CUDA_CHECK(hipMemcpy(greenSumShadowArray, deviceGreenSumShadowArray,
                      imageSize * sizeof(float),
                      hipMemcpyDeviceToHost));
CUDA_CHECK(hipMemcpy(blueSumShadowArray, deviceBlueSumShadowArray,
                      imageSize * sizeof(float),
                      hipMemcpyDeviceToHost));
CUDA_CHECK(hipMemcpy(redSumLightArray, deviceRedSumLightArray,
                      imageSize * sizeof(float),
                      hipMemcpyDeviceToHost));
CUDA_CHECK(hipMemcpy(greenSumLightArray, deviceGreenSumLightArray,
                      imageSize * sizeof(float),
                      hipMemcpyDeviceToHost));
CUDA_CHECK(hipMemcpy(blueSumLightArray, deviceBlueSumLightArray,
                      imageSize * sizeof(float),
                      hipMemcpyDeviceToHost));  
CUDA_CHECK(hipMemcpy(erodedSumShadowArray, deviceErodedSumShadowArray,
                      imageSize * sizeof(float),
                      hipMemcpyDeviceToHost));     
CUDA_CHECK(hipMemcpy(erodedSumLightArray, deviceErodedSumLightArray,
                      imageSize * sizeof(float),
                      hipMemcpyDeviceToHost));                
CUDA_CHECK(hipDeviceSynchronize());
wbTime_stop(Copy, "Copying output memory to the CPU");


  wbTime_start(GPU, "Allocating GPU memory.");
  CUDA_CHECK( hipMalloc((void **)&deviceFinalImage, imageSize * sizeof(float))); 
  CUDA_CHECK( hipMalloc((void **)&deviceSmoothMask, imageSize * sizeof(float)));      
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(GPU, "Allocating GPU memory.");


  finalImage = (float *)malloc(imageSize * sizeof(float));
  // zero out bins
  CUDA_CHECK(hipMemset(deviceFinalImage, 0.0, imageSize * sizeof(float)));
  CUDA_CHECK(hipGetLastError());
  // Launch calculate_rgb_ratio kernel on the eroded shadow array and calculates the final image
    calculate_final_image<<<gridDim2, blockDim2>>>(
    deviceRedSumShadowArray, deviceGreenSumShadowArray,deviceBlueSumShadowArray,
    deviceRedSumLightArray, deviceGreenSumLightArray,deviceBlueSumLightArray,
    deviceErodedSumShadowArray,deviceErodedSumLightArray,
    deviceRgbImage, deviceSmoothMask, deviceFinalImage,
    imageWidth, imageHeight, NUM_CHANNELS);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
   

  //@@ Copy the GPU memory back to the CPU here
  wbTime_start(Copy, "Copying output memory to the CPU");
  CUDA_CHECK(hipMemcpy(finalImage, deviceFinalImage,
                        imageSize * sizeof(float),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(Copy, "Copying output memory to the CPU");

  // //@@ Free the GPU memory here
  wbTime_start(Copy, "Freeing GPU Memory");
  CUDA_CHECK(hipFree(deviceRgbImage));
  CUDA_CHECK(hipFree(deviceRedShadowArray));
  CUDA_CHECK(hipFree(deviceGreenShadowArray));
  CUDA_CHECK(hipFree(deviceBlueShadowArray));
  CUDA_CHECK(hipFree(deviceRedLightArray));
  CUDA_CHECK(hipFree(deviceGreenLightArray));
  CUDA_CHECK(hipFree(deviceBlueLightArray));
  CUDA_CHECK(hipFree(deviceErodedShadowMask));
  CUDA_CHECK(hipFree(deviceErodedLightMask));
  CUDA_CHECK(hipFree(deviceRedSumShadowArray));
  CUDA_CHECK(hipFree(deviceGreenSumShadowArray));
  CUDA_CHECK(hipFree(deviceBlueSumShadowArray));
  CUDA_CHECK(hipFree(deviceRedSumLightArray)); 
  CUDA_CHECK(hipFree(deviceGreenSumLightArray));
  CUDA_CHECK(hipFree(deviceBlueSumLightArray));
  CUDA_CHECK(hipFree(deviceErodedSumShadowArray));
  CUDA_CHECK(hipFree(deviceErodedSumLightArray));
  CUDA_CHECK(hipFree(deviceSmoothMask));
  CUDA_CHECK(hipFree(deviceFinalImage));
  wbTime_stop(Copy, "Freeing GPU Memory");

}