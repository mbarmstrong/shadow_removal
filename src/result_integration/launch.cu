#include <cv.h>
#include "kernel.cu"


void launch_result_integration(wbImage_t& image, unsigned int* bins) {

  float *rgbImage;
  float *greyShadowMask;
  float *greyLightMask; 
  float *redShadowArray;
  float *greenShadowArray;
  float *blueShadowArray;
  float *redLightArray;
  float *greenLightArray;
  float *blueLightArray;
  float *erodedShadowArray;
  float *erodedLightArray;
  float *redSumShadowArray;
  float *greenSumShadowArray;
  float *blueSumShadowArray;
  float *redSumLightArray; 
  float *greenSumLightArray;
  float *blueSumLightArray;
  float *erodedSumShadowArray;
  float *erodedSumShadowArray;
  float *smoothMask;
  float *finalImage;
  float *deviceRgbImage;
  float *deviceGreyShadowMask;
  float *deviceGreyLightMask; 
  float *deviceRedShadowArray;
  float *deviceGreenShadowArray;
  float *deviceBlueShadowArray;
  float *deviceRedLightArray;
  float *deviceGreenLightArray;
  float *deviceBlueLightArray;
  float *deviceErodedShadowArray;
  float *deviceErodedLightArray;
  float *deviceRedSumShadowArray;
  float *deviceGreenSumShadowArray;
  float *deviceBlueSumShadowArray;
  float *deviceRedSumLightArray; 
  float *deviceGreenSumLightArray;
  float *deviceBlueSumLightArray;
  float *deviceErodedSumShadowArray;
  float *deviceErodedSumShadowArray;
  float *deviceSmoothMask;
  float *deviceFinalImage;
  int redRatio;
  int greenRatio;
  int blueRatio;
  int imageChannels;
  int imageWidth;
  int imageHeight;
  int imageSize;
  int deviceImageChannels;
  int deviceImageWidth;
  int deviceImageHeight;

  rgbImage  = wbImage_getData(image);
  
  imageWidth  = wbImage_getWidth(image);
  imageHeight = wbImage_getHeight(image);
  imageChannels = wbImage_getChannels(image);


  //@@ Allocate GPU memory here
  wbTime_start(GPU, "Allocating GPU memory.");
  CUDA_CHECK( hipMalloc((void **)&deviceImageChannels,imageChannels * sizeof(int)));
  CUDA_CHECK( hipMalloc((void **)&deviceImageWidth,imageWidth * sizeof(int)));
  CUDA_CHECK( hipMalloc((void **)&deviceImageHeight,imageHeight * sizeof(int)) );
  CUDA_CHECK( hipDeviceSynchronize() );
  wbTime_stop(GPU, "Allocating GPU memory.");

  //@@ Copy memory to the GPU here
  wbTime_start(GPU, "Copying input memory to the GPU.");
  CUDA_CHECK(hipMemcpy(deviceRgbImage, rgbImage,
                        rgbImage * sizeof(float),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  // zero out bins
  CUDA_CHECK(hipMemset(deviceBins, 0, NUM_BINS * sizeof(unsigned int)));
  
  // Launch multiple_rgbImage_byMask kernel on the bins
  {
    dim3 blockDim(512), gridDim(30);
    multiple_rgbImage_byMask<<<gridDim, blockDim>>>(
      deviceRgbImage, deviceGreyShadowMask, 
      deviceGreyLightMask, deviceRedShadowArray,deviceGreenShadowArray,deviceBlueShadowArray,
      deviceRedLightArray,deviceGreenLightArray,deviceBlueLightArray,int width, int height, int numChannels);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
  }
  /@@ Copy the GPU memory back to the CPU here
  wbTime_start(Copy, "Copying output memory to the CPU");
  CUDA_CHECK(hipMemcpy(redShadowArray, deviceRedShadowArray,
                        deviceRedShadowArray * sizeof(float),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(greenShadowArray, deviceGreenShadowArray,
                        deviceGreenShadowArray * sizeof(float),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(blueShadowArray, deviceBlueShadowArray,
                        deviceBlueShadowArray * sizeof(float),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(redLightArray, deviceRedLightArray,
                        deviceRedLightArray * sizeof(float),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(greenShadowArray, deviceGreenLightArray,
                        deviceGreenLightArray * sizeof(float),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(blueLightArray, deviceBlueLightArray,
                        deviceBlueLightArray * sizeof(float),
                        hipMemcpyDeviceToHost));                       
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(Copy, "Copying output memory to the CPU");

  // Launch sum_up_arrays kernel on the light and shadow arrays for each channel
  {
    dim3 blockDim(512), gridDim(30);
    sum_up_arrays_by_reduction<<<gridDim, blockDim>>>(
        deviceRedShadowArray,deviceRedSumShadowArray, int width, int height);
      CUDA_CHECK(hipGetLastError());
      CUDA_CHECK(hipDeviceSynchronize());
  }
   // Launch sum_up_arrays kernel on the light and shadow arrays for each channel
   {
    dim3 blockDim(512), gridDim(30);
    sum_up_arrays_by_reduction<<<gridDim, blockDim>>>(
        deviceGreenShadowArray,deviceGreenSumShadowArray, int width, int height);
      CUDA_CHECK(hipGetLastError());
      CUDA_CHECK(hipDeviceSynchronize());
  }
   // Launch sum_up_arrays kernel on the shadow arrays for each channel
   {
    dim3 blockDim(512), gridDim(30);
    sum_up_arrays_by_reduction<<<gridDim, blockDim>>>(
        deviceBlueShadowArray,deviceBlueSumShadowArray, int width, int height);
      CUDA_CHECK(hipGetLastError());
      CUDA_CHECK(hipDeviceSynchronize());
  }
   // Launch sum_up_arrays kernel on the light arrays for each channel
   {
    dim3 blockDim(512), gridDim(30);
    sum_up_arrays_by_reduction<<<gridDim, blockDim>>>(
        deviceRedLightArray,deviceRedSumLightArray, int width, int height);
      CUDA_CHECK(hipGetLastError());
      CUDA_CHECK(hipDeviceSynchronize());
  }
  {
    dim3 blockDim(512), gridDim(30);
    sum_up_arrays_by_reduction<<<gridDim, blockDim>>>(
          deviceGreenLightArray,deviceGreenSumLightArray, int width, int height);
      CUDA_CHECK(hipGetLastError());
      CUDA_CHECK(hipDeviceSynchronize());
  }
  {
    dim3 blockDim(512), gridDim(30);
    sum_up_arrays_by_reduction<<<gridDim, blockDim>>>(
          deviceBlueLightArray,deviceBlueSumLightArray, int width, int height);
      CUDA_CHECK(hipGetLastError());
      CUDA_CHECK(hipDeviceSynchronize());
  }
  // Launch sum_up_arrays kernel on the eroded shadow array
  {
    dim3 blockDim(512), gridDim(30);
    sum_up_arrays_by_reduction<<<gridDim, blockDim>>>(
      deviceErodedShadowArray,deviceErodedSumShadowArray, int width, int height);
      CUDA_CHECK(hipGetLastError());
      CUDA_CHECK(hipDeviceSynchronize());
  } 
  // Launch sum_up_arrays kernel on the eroded light array
  {
    dim3 blockDim(512), gridDim(30);
    sum_up_arrays_by_reduction<<<gridDim, blockDim>>>(
      deviceErodedLightArray,deviceErodedSumLightArray, int width, int height);
      CUDA_CHECK(hipGetLastError());
      CUDA_CHECK(hipDeviceSynchronize());
  } 

  /@@ Copy the GPU memory back to the CPU here
  wbTime_start(Copy, "Copying output memory to the CPU");
  CUDA_CHECK(hipMemcpy(redSumShadowArray, deviceRedSumShadowArray,
                        deviceRedSumShadowArray * sizeof(float),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(greenSumShadowArray, deviceGreenSumShadowArray,
                        deviceGreenSumShadowArray * sizeof(float),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(blueSumShadowArray, deviceBlueSumShadowArray,
                        deviceBlueSumShadowArray * sizeof(float),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(redSumLightArray, deviceRedSumLightArray,
                        deviceRedSumShadowArray * sizeof(float),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(greenSumLightArray, deviceGreenSumLightArray,
                        deviceGreenSumShadowArray * sizeof(float),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(blueSumLightArray, deviceBlueSumLightArray,
                        deviceBlueSumLightArray * sizeof(float),
                        hipMemcpyDeviceToHost));  
  CUDA_CHECK(hipMemcpy(erodedSumShadowArray, deviceErodedSumShadowArray,
                        deviceErodedSumShadowArray * sizeof(float),
                        hipMemcpyDeviceToHost));                    
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(Copy, "Copying output memory to the CPU");

  // Launch calculate_rgb_ratio kernel on the eroded shadow array
  {
    dim3 blockDim(512), gridDim(30);
    calculate_rgb_ratio<<<gridDim, blockDim>>>(
    redSumShadowArray, greenSumShadowArray,blueSumShadowArray,
    redSumLightArray, greenSumLightArray,blueSumLightArray,
    erodedSumShadowArray,erodedSumShadowArray,
    rgbImage, smoothMask, finalImage,
    width, height, numChannels);
      CUDA_CHECK(hipGetLastError());
      CUDA_CHECK(hipDeviceSynchronize());
  } 

  //@@ Copy the GPU memory back to the CPU here
  wbTime_start(Copy, "Copying output memory to the CPU");
  CUDA_CHECK(hipMemcpy(finalImage, deviceFinalImage,
                        deviceFinalImage * sizeof(float),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(Copy, "Copying output memory to the CPU");

  //@@ Free the GPU memory here
  wbTime_start(GPU, "Freeing GPU Memory");
  CUDA_CHECK(hipFree(deviceImageData));
  CUDA_CHECK(hipFree(deviceBins));
  wbTime_stop(GPU, "Freeing GPU Memory");

  imShow(finalImage);

}