#include "hip/hip_runtime.h"
#include <wb.h>
#include "kernel.cu"
#include "../globals.h"


void launch_result_integration(float *rgbImage,unsigned char *erodedShadowMask,unsigned char *erodedLightMask, 
  float *smoothMask,unsigned char *finalImage,int imageWidth, int imageHeight) {
  
  unsigned char *redShadowArray;
  unsigned char *greenShadowArray;
  unsigned char *blueShadowArray;
  unsigned char *redLightArray;
  unsigned char *greenLightArray;
  unsigned char *blueLightArray;
  unsigned char *redSumShadowArray;
  unsigned char *greenSumShadowArray;
  unsigned char *blueSumShadowArray;
  unsigned char *redSumLightArray; 
  unsigned char *greenSumLightArray;
  unsigned char *blueSumLightArray;
  unsigned char *erodedSumShadowArray;
  unsigned char *erodedSumLightArray;
  float *deviceRgbImage;
  unsigned char *deviceRedShadowArray;
  unsigned char *deviceGreenShadowArray;
  unsigned char *deviceBlueShadowArray;
  unsigned char *deviceRedLightArray;
  unsigned char *deviceGreenLightArray;
  unsigned char *deviceBlueLightArray;
  unsigned char *deviceErodedShadowMask;
  unsigned char *deviceErodedLightMask;
  unsigned char *deviceRedSumShadowArray;
  unsigned char *deviceGreenSumShadowArray;
  unsigned char *deviceBlueSumShadowArray;
  unsigned char *deviceRedSumLightArray; 
  unsigned char *deviceGreenSumLightArray;
  unsigned char *deviceBlueSumLightArray;
  unsigned char *deviceErodedSumShadowArray;
  unsigned char *deviceErodedSumLightArray;
  float *deviceSmoothMask;
  unsigned char *deviceFinalImage;

  int imageSize = imageHeight * imageWidth;
  int n_threads = 16;

  wbTime_start(GPU, "Allocating GPU memory.");
  CUDA_CHECK( hipMalloc((void **)&deviceRgbImage, imageSize * sizeof(float)));
  CUDA_CHECK( hipMalloc((void **)&deviceErodedShadowMask, imageSize * sizeof(unsigned char)));
  CUDA_CHECK( hipMalloc((void **)&deviceErodedLightMask, imageSize * sizeof(unsigned char)));
  CUDA_CHECK( hipMalloc((void **)&deviceRedShadowArray, imageSize * sizeof(unsigned char)));
  CUDA_CHECK( hipMalloc((void **)&deviceGreenShadowArray, imageSize * sizeof(unsigned char)));
  CUDA_CHECK( hipMalloc((void **)&deviceBlueShadowArray, imageSize * sizeof(unsigned char)));
  CUDA_CHECK( hipMalloc((void **)&deviceRedLightArray, imageSize * sizeof(unsigned char)));
  CUDA_CHECK( hipMalloc((void **)&deviceGreenLightArray, imageSize * sizeof(unsigned char)));
  CUDA_CHECK( hipMalloc((void **)&deviceBlueLightArray, imageSize * sizeof(unsigned char)));     
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(GPU, "Allocating GPU memory.");

  // Copy memory to the GPU here
  wbTime_start(GPU, "Copying input memory to the GPU.");
  CUDA_CHECK(hipMemcpy(deviceRgbImage, rgbImage, imageSize * sizeof(float),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(deviceErodedShadowMask, erodedShadowMask, imageSize * sizeof(unsigned char),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(deviceErodedLightMask, erodedLightMask, imageSize * sizeof(unsigned char),
                        hipMemcpyHostToDevice));    
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(GPU, "Copying input memory to the GPU.");
  
  // Launch multiple_rgbImage_byMask kernel on the bins
  {
    dim3 gridDim(ceil((float)imageWidth/(float)n_threads),ceil((float)imageHeight/(float)n_threads));
    dim3 blockDim(n_threads,n_threads);
    multiply_rgbImage_byMask<<<gridDim, blockDim>>>(
      deviceRgbImage, deviceErodedShadowMask, 
      deviceErodedLightMask, deviceRedShadowArray,deviceGreenShadowArray,deviceBlueShadowArray,
      deviceRedLightArray,deviceGreenLightArray,deviceBlueLightArray,imageWidth,imageHeight, NUM_CHANNELS);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
  }

  redShadowArray = (unsigned char *)malloc(imageSize * sizeof(unsigned char));
  blueShadowArray = (unsigned char *)malloc(imageSize * sizeof(unsigned char));
  greenShadowArray = (unsigned char *)malloc(imageSize * sizeof(unsigned char));
  redLightArray = (unsigned char *)malloc(imageSize * sizeof(unsigned char));
  greenLightArray = (unsigned char *)malloc(imageSize * sizeof(unsigned char));
  blueLightArray = (unsigned char *)malloc(imageSize * sizeof(unsigned char));
 // Copy the GPU memory back to the CPU here
  wbTime_start(Copy, "Copying output memory to the CPU");
  CUDA_CHECK(hipMemcpy(redShadowArray, deviceRedShadowArray,
                        imageSize * sizeof(unsigned char ),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(greenShadowArray, deviceGreenShadowArray,
                        imageSize * sizeof(unsigned char ),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(blueShadowArray, deviceBlueShadowArray,
                        imageSize * sizeof(unsigned char ),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(redLightArray, deviceRedLightArray,
                        imageSize * sizeof(unsigned char ),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(greenLightArray, deviceGreenLightArray,
                        imageSize * sizeof(unsigned char ),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(blueLightArray, deviceBlueLightArray,
                        imageSize * sizeof(unsigned char ),
                        hipMemcpyDeviceToHost));                       
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(Copy, "Copying output memory to the CPU");


  wbTime_start(GPU, "Allocating GPU memory.");
  CUDA_CHECK( hipMalloc((void **)&deviceRedSumShadowArray, imageSize * sizeof(unsigned char)));    
  CUDA_CHECK( hipMalloc((void **)&deviceGreenSumShadowArray, imageSize * sizeof(unsigned char)));    
  CUDA_CHECK( hipMalloc((void **)&deviceBlueSumShadowArray, imageSize * sizeof(unsigned char)));    
  CUDA_CHECK( hipMalloc((void **)&deviceRedSumLightArray, imageSize * sizeof(unsigned char)));    
  CUDA_CHECK( hipMalloc((void **)&deviceGreenSumLightArray, imageSize * sizeof(unsigned char)));
  CUDA_CHECK( hipMalloc((void **)&deviceBlueSumLightArray, imageSize * sizeof(unsigned char))); 
  CUDA_CHECK( hipMalloc((void **)&deviceErodedSumLightArray, imageSize * sizeof(unsigned char))); 
  CUDA_CHECK( hipMalloc((void **)&deviceErodedSumShadowArray, imageSize * sizeof(unsigned char)));       
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(GPU, "Allocating GPU memory.");

  // Launch sum_up_arrays kernel on the light and shadow arrays for each channel
  {
    dim3 gridDim(ceil((float)imageWidth/(float)n_threads),ceil((float)imageHeight/(float)n_threads));
    dim3 blockDim(n_threads,n_threads);
    sum_up_arrays_by_reduction<<<gridDim, blockDim>>>(
        deviceRedShadowArray,deviceRedSumShadowArray,imageSize);
      CUDA_CHECK(hipGetLastError());
      CUDA_CHECK(hipDeviceSynchronize());
  }
   // Launch sum_up_arrays kernel on the light and shadow arrays for each channel
   {
    dim3 blockDim(8,8), gridDim(1,1);
    sum_up_arrays_by_reduction<<<gridDim, blockDim>>>(
        deviceGreenShadowArray,deviceGreenSumShadowArray,imageSize);
      CUDA_CHECK(hipGetLastError());
      CUDA_CHECK(hipDeviceSynchronize());
  }
   // Launch sum_up_arrays kernel on the shadow arrays for each channel
   {
    dim3 gridDim(ceil((float)imageWidth/(float)n_threads),ceil((float)imageHeight/(float)n_threads));
    dim3 blockDim(n_threads,n_threads);
    sum_up_arrays_by_reduction<<<gridDim, blockDim>>>(
        deviceBlueShadowArray,deviceBlueSumShadowArray,imageSize);
      CUDA_CHECK(hipGetLastError());
      CUDA_CHECK(hipDeviceSynchronize());
  }
   // Launch sum_up_arrays kernel on the light arrays for each channel
   {
    dim3 gridDim(ceil((float)imageWidth/(float)n_threads),ceil((float)imageHeight/(float)n_threads));
    dim3 blockDim(n_threads,n_threads);
    sum_up_arrays_by_reduction<<<gridDim, blockDim>>>(
        deviceRedLightArray,deviceRedSumLightArray,imageSize);
      CUDA_CHECK(hipGetLastError());
      CUDA_CHECK(hipDeviceSynchronize());
  }
  {
    dim3 gridDim(ceil((float)imageWidth/(float)n_threads),ceil((float)imageHeight/(float)n_threads));
    dim3 blockDim(n_threads,n_threads);
    sum_up_arrays_by_reduction<<<gridDim, blockDim>>>(
          deviceGreenLightArray,deviceGreenSumLightArray,imageSize);
      CUDA_CHECK(hipGetLastError());
      CUDA_CHECK(hipDeviceSynchronize());
  }
  {
    dim3 gridDim(ceil((float)imageWidth/(float)n_threads),ceil((float)imageHeight/(float)n_threads));
    dim3 blockDim(n_threads,n_threads);
    sum_up_arrays_by_reduction<<<gridDim, blockDim>>>(
          deviceBlueLightArray,deviceBlueSumLightArray,imageSize);
      CUDA_CHECK(hipGetLastError());
      CUDA_CHECK(hipDeviceSynchronize());
  }
  // Launch sum_up_arrays kernel on the eroded shadow array
  {
    dim3 gridDim(ceil((float)imageWidth/(float)n_threads),ceil((float)imageHeight/(float)n_threads));
    dim3 blockDim(n_threads,n_threads);
    sum_up_arrays_by_reduction<<<gridDim, blockDim>>>(
      deviceErodedShadowMask,deviceErodedSumShadowArray,imageSize);
      CUDA_CHECK(hipGetLastError());
      CUDA_CHECK(hipDeviceSynchronize());
  } 
  // Launch sum_up_arrays kernel on the eroded light array
  {
    dim3 gridDim(ceil((float)imageWidth/(float)n_threads),ceil((float)imageHeight/(float)n_threads));
    dim3 blockDim(n_threads,n_threads);
    sum_up_arrays_by_reduction<<<gridDim, blockDim>>>(
      deviceErodedLightMask,deviceErodedSumLightArray,imageSize);
      CUDA_CHECK(hipGetLastError());
      CUDA_CHECK(hipDeviceSynchronize());
  } 

  redSumShadowArray = (unsigned char *)malloc(imageSize * sizeof(unsigned char));
  greenSumShadowArray = (unsigned char *)malloc(imageSize * sizeof(unsigned char));
  blueSumShadowArray = (unsigned char *)malloc(imageSize * sizeof(unsigned char));
  redSumLightArray = (unsigned char *)malloc(imageSize * sizeof(unsigned char));
  greenSumLightArray = (unsigned char *)malloc(imageSize * sizeof(unsigned char));
  blueSumLightArray = (unsigned char *)malloc(imageSize * sizeof(unsigned char));
  erodedSumShadowArray = (unsigned char *)malloc(imageSize * sizeof(unsigned char));
  erodedSumLightArray = (unsigned char *)malloc(imageSize * sizeof(unsigned char));

  // Copy the GPU memory back to the CPU here
  wbTime_start(Copy, "Copying output memory to the CPU");
  CUDA_CHECK(hipMemcpy(redSumShadowArray, deviceRedSumShadowArray,
                        imageSize * sizeof(unsigned char),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(greenSumShadowArray, deviceGreenSumShadowArray,
                        imageSize * sizeof(unsigned char),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(blueSumShadowArray, deviceBlueSumShadowArray,
                        imageSize * sizeof(unsigned char),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(redSumLightArray, deviceRedSumLightArray,
                        imageSize * sizeof(unsigned char),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(greenSumLightArray, deviceGreenSumLightArray,
                        imageSize * sizeof(unsigned char),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(blueSumLightArray, deviceBlueSumLightArray,
                        imageSize * sizeof(unsigned char),
                        hipMemcpyDeviceToHost));  
  CUDA_CHECK(hipMemcpy(erodedSumShadowArray, deviceErodedSumShadowArray,
                        imageSize * sizeof(unsigned char),
                        hipMemcpyDeviceToHost));     
  CUDA_CHECK(hipMemcpy(erodedSumLightArray, deviceErodedSumLightArray,
                        imageSize * sizeof(unsigned char),
                        hipMemcpyDeviceToHost));                
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(Copy, "Copying output memory to the CPU");


  wbTime_start(GPU, "Allocating GPU memory.");
  CUDA_CHECK( hipMalloc((void **)&deviceFinalImage, imageSize * sizeof(unsigned char))); 
  CUDA_CHECK( hipMalloc((void **)&deviceSmoothMask, imageSize * sizeof(float)));      
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(GPU, "Allocating GPU memory.");

  // zero out bins
    CUDA_CHECK(hipMemset(deviceFinalImage, 0.0, imageSize * sizeof(unsigned char)));
  // Launch calculate_rgb_ratio kernel on the eroded shadow array and calculates the final image
  {
    dim3 blockDim(8,8), gridDim(1,1);
    calculate_final_image<<<gridDim, blockDim>>>(
    deviceRedSumShadowArray, deviceGreenSumShadowArray,deviceBlueSumShadowArray,
    deviceRedSumLightArray, deviceGreenSumLightArray,deviceBlueSumLightArray,
    deviceErodedSumShadowArray,deviceErodedSumLightArray,
    deviceRgbImage, deviceSmoothMask, deviceFinalImage,
    imageWidth, imageHeight, NUM_CHANNELS);
      CUDA_CHECK(hipGetLastError());
      CUDA_CHECK(hipDeviceSynchronize());
  } 

  //@@ Copy the GPU memory back to the CPU here
  wbTime_start(Copy, "Copying output memory to the CPU");
  CUDA_CHECK(hipMemcpy(finalImage, deviceFinalImage,
                        imageSize * sizeof(unsigned char),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(Copy, "Copying output memory to the CPU");

  //@@ Free the GPU memory here
  wbTime_start(GPU, "Freeing GPU Memory");

  CUDA_CHECK(hipFree(deviceRgbImage));
  CUDA_CHECK(hipFree(deviceRedShadowArray));
  CUDA_CHECK(hipFree(deviceGreenShadowArray));
  CUDA_CHECK(hipFree(deviceBlueShadowArray));
  CUDA_CHECK(hipFree(deviceRedLightArray));
  CUDA_CHECK(hipFree(deviceGreenLightArray));
  CUDA_CHECK(hipFree(deviceBlueLightArray));
  CUDA_CHECK(hipFree(deviceErodedShadowMask));
  CUDA_CHECK(hipFree(deviceErodedLightMask));
  CUDA_CHECK(hipFree(deviceRedSumShadowArray));
  CUDA_CHECK(hipFree(deviceGreenSumShadowArray));
  CUDA_CHECK(hipFree(deviceBlueSumShadowArray));
  CUDA_CHECK(hipFree(deviceRedSumLightArray)); 
  CUDA_CHECK(hipFree(deviceGreenSumLightArray));
  CUDA_CHECK(hipFree(deviceBlueSumLightArray));
  CUDA_CHECK(hipFree(deviceErodedSumShadowArray));
  CUDA_CHECK(hipFree(deviceErodedSumLightArray));
  CUDA_CHECK(hipFree(deviceSmoothMask));
  CUDA_CHECK(hipFree(deviceFinalImage));
  wbTime_stop(GPU, "Freeing GPU Memory");

}