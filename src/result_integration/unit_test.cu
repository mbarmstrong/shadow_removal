
#include <wb.h>
#include "kernel.cu"
#include "../globals.h"

void unit_test( float *rgbImage,unsigned char *erodedShadowMask,unsigned char *erodedLightMask, float *smoothMask,int imageWidth, int imageHeight) {

  float *redShadowArray;
  float *greenShadowArray;
  float *blueShadowArray;
  float *redLightArray;
  float *greenLightArray;
  float *blueLightArray;
  float *redSumShadowArray;
  float *greenSumShadowArray;
  float *blueSumShadowArray;
  float *redSumLightArray; 
  float *greenSumLightArray;
  float *blueSumLightArray;
  float *erodedSumShadowArray;
  float *erodedSumLightArray;
  float *deviceRgbImage;
  float *deviceRedShadowArray;
  float *deviceGreenShadowArray;
  float *deviceBlueShadowArray;
  float *deviceRedLightArray;
  float *deviceGreenLightArray;
  float *deviceBlueLightArray;
  unsigned char *deviceErodedShadowMask;
  unsigned char *deviceErodedLightMask;
  float *deviceRedSumShadowArray;
  float *deviceGreenSumShadowArray;
  float *deviceBlueSumShadowArray;
  float *deviceRedSumLightArray; 
  float *deviceGreenSumLightArray;
  float *deviceBlueSumLightArray;
  float *deviceErodedSumShadowArray;
  float *deviceErodedSumLightArray;
  float *deviceRedSumShadowArray_interm;
  float *deviceGreenSumShadowArray_interm;
  float *deviceBlueSumShadowArray_interm;
  float *deviceRedSumLightArray_interm; 
  float *deviceGreenSumLightArray_interm;
  float *deviceBlueSumLightArray_interm;
  float *deviceErodedSumShadowArray_interm;
  float *deviceErodedSumLightArray_interm;
  float *deviceSmoothMask;
  float *deviceFinalImage;

  float *finalImage;

  int imageSize = imageHeight * imageWidth;

  wbTime_start(GPU, "Allocating GPU memory.");
  CUDA_CHECK( hipMalloc((void **)&deviceRgbImage, imageSize * sizeof(float)));
  CUDA_CHECK( hipMalloc((void **)&deviceSmoothMask, imageSize * sizeof(float)));
  CUDA_CHECK( hipMalloc((void **)&deviceErodedShadowMask, imageSize * sizeof(unsigned char)));
  CUDA_CHECK( hipMalloc((void **)&deviceErodedLightMask, imageSize * sizeof(unsigned char)));
  CUDA_CHECK( hipMalloc((void **)&deviceRedShadowArray, imageSize * sizeof(float)));
  CUDA_CHECK( hipMalloc((void **)&deviceGreenShadowArray, imageSize * sizeof(float)));
  CUDA_CHECK( hipMalloc((void **)&deviceBlueShadowArray, imageSize * sizeof(float)));
  CUDA_CHECK( hipMalloc((void **)&deviceRedLightArray, imageSize * sizeof(float)));
  CUDA_CHECK( hipMalloc((void **)&deviceGreenLightArray, imageSize * sizeof(float)));
  CUDA_CHECK( hipMalloc((void **)&deviceBlueLightArray, imageSize * sizeof(float)));     
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(GPU, "Allocating GPU memory.");

  // Copy memory to the GPU here
  wbTime_start(GPU, "Copying input memory to the GPU.");
  CUDA_CHECK(hipMemcpy(deviceRgbImage, rgbImage, imageSize * sizeof(float),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(deviceSmoothMask, smoothMask, imageSize * sizeof(float),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(deviceErodedShadowMask, erodedShadowMask, imageSize * sizeof(unsigned char),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(deviceErodedLightMask, erodedLightMask, imageSize * sizeof(unsigned char),
                        hipMemcpyHostToDevice));    
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  printf("\nGray Shadow Mask:\t");
  print_image(erodedShadowMask,imageWidth,imageHeight);

  printf("\nGray Light Mask:\t");
  print_image(erodedLightMask,imageWidth,imageHeight);
  
  // Launch multiple_rgbImage_byMask kernel on the bins
  {
    dim3 blockDim(8,8), gridDim(1,1);
    multiply_rgbImage_byMask<<<gridDim, blockDim>>>(
      deviceRgbImage, deviceErodedShadowMask, 
      deviceErodedLightMask, deviceRedShadowArray,deviceGreenShadowArray,deviceBlueShadowArray,
      deviceRedLightArray,deviceGreenLightArray,deviceBlueLightArray,imageWidth,imageHeight, NUM_CHANNELS);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
  }

  redShadowArray = (float *)malloc(imageSize * sizeof(float));
  blueShadowArray = (float *)malloc(imageSize * sizeof(float));
  greenShadowArray = (float *)malloc(imageSize * sizeof(float));
  redLightArray = (float *)malloc(imageSize * sizeof(float));
  greenLightArray = (float *)malloc(imageSize * sizeof(float));
  blueLightArray = (float *)malloc(imageSize * sizeof(float));
 // Copy the GPU memory back to the CPU here
  wbTime_start(Copy, "Copying output memory to the CPU");
  CUDA_CHECK(hipMemcpy(redShadowArray, deviceRedShadowArray,
                        imageSize * sizeof(float),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(greenShadowArray, deviceGreenShadowArray,
                        imageSize * sizeof(float),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(blueShadowArray, deviceBlueShadowArray,
                        imageSize * sizeof(float),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(redLightArray, deviceRedLightArray,
                        imageSize * sizeof(float),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(greenLightArray, deviceGreenLightArray,
                        imageSize * sizeof(float),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(blueLightArray, deviceBlueLightArray,
                        imageSize * sizeof(float),
                        hipMemcpyDeviceToHost));                       
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(Copy, "Copying output memory to the CPU");

  printf("\nRed Shadow Array:\t");
  print_image(redShadowArray,imageWidth,imageHeight);
  printf("\nGreen Shadow Array:\t");
  print_image(greenShadowArray,imageWidth,imageHeight);
  printf("\nBlue Shadow Array:\t");
  print_image(blueShadowArray,imageWidth,imageHeight);
  printf("\nRed Light Array:\t");
  print_image(redLightArray,imageWidth,imageHeight);
  printf("\nGreen Light Array:\t");
  print_image(greenLightArray,imageWidth,imageHeight);
  printf("\nBlue Light Array:\t");
  print_image(blueLightArray,imageWidth,imageHeight);


  wbTime_start(GPU, "Allocating GPU memory.");
  CUDA_CHECK( hipMalloc((void **)&deviceRedSumShadowArray, imageSize * sizeof(float)));   
  CUDA_CHECK( hipMalloc((void **)&deviceRedSumShadowArray_interm, imageSize * sizeof(float)));   
  CUDA_CHECK( hipMalloc((void **)&deviceGreenSumShadowArray, imageSize * sizeof(float)));   
  CUDA_CHECK( hipMalloc((void **)&deviceGreenSumShadowArray_interm, imageSize * sizeof(float))); 
  CUDA_CHECK( hipMalloc((void **)&deviceBlueSumShadowArray, imageSize * sizeof(float)));   
  CUDA_CHECK( hipMalloc((void **)&deviceBlueSumShadowArray_interm, imageSize * sizeof(float)));  
  CUDA_CHECK( hipMalloc((void **)&deviceRedSumLightArray, imageSize * sizeof(float)));   
  CUDA_CHECK( hipMalloc((void **)&deviceRedSumLightArray_interm, imageSize * sizeof(float)));  
  CUDA_CHECK( hipMalloc((void **)&deviceGreenSumLightArray, imageSize * sizeof(float)));
  CUDA_CHECK( hipMalloc((void **)&deviceGreenSumLightArray_interm, imageSize * sizeof(float)));
  CUDA_CHECK( hipMalloc((void **)&deviceBlueSumLightArray, imageSize * sizeof(float))); 
  CUDA_CHECK( hipMalloc((void **)&deviceBlueSumLightArray_interm, imageSize * sizeof(float))); 
  CUDA_CHECK( hipMalloc((void **)&deviceErodedSumLightArray, imageSize * sizeof(float))); 
  CUDA_CHECK( hipMalloc((void **)&deviceErodedSumLightArray_interm, imageSize * sizeof(float))); 
  CUDA_CHECK( hipMalloc((void **)&deviceErodedSumShadowArray, imageSize * sizeof(float)));  
  CUDA_CHECK( hipMalloc((void **)&deviceErodedSumShadowArray_interm, imageSize * sizeof(float)));        
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(GPU, "Allocating GPU memory.");

  // Launch sum_up_arrays kernel on the light and shadow arrays for each channel
    const int maxThreadsPerBlock = 16;
    int threads = maxThreadsPerBlock;
    int blocks = imageSize / maxThreadsPerBlock;
    sum_up_arrays_by_reduction<<<blocks, threads, threads * sizeof(float)>>>(
        deviceRedShadowArray,deviceRedSumShadowArray_interm,imageSize);
        CUDA_CHECK(hipGetLastError());
      CUDA_CHECK(hipDeviceSynchronize());
    threads = blocks; // launch one thread for each block in prev step
    blocks = 1;
    sum_up_arrays_by_reduction<<<blocks, threads, threads * sizeof(float)>>>(
    deviceRedSumShadowArray_interm,deviceRedSumShadowArray,imageSize);  
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
   // Launch sum_up_arrays kernel on the light and shadow arrays for each channel
    threads = maxThreadsPerBlock;
    blocks = imageSize / maxThreadsPerBlock;
    sum_up_arrays_by_reduction<<<blocks, threads, threads * sizeof(float)>>>(
        deviceGreenShadowArray,deviceGreenSumShadowArray_interm,imageSize);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());    
    threads = blocks; // launch one thread for each block in prev step
    blocks = 1;
    sum_up_arrays_by_reduction<<<blocks, threads, threads * sizeof(float)>>>(
    deviceGreenSumShadowArray_interm,deviceGreenSumShadowArray,imageSize); 
      CUDA_CHECK(hipGetLastError());
      CUDA_CHECK(hipDeviceSynchronize());

   // Launch sum_up_arrays kernel on the shadow arrays for each channel
    threads = maxThreadsPerBlock;
    blocks = imageSize / maxThreadsPerBlock;
    sum_up_arrays_by_reduction<<<blocks, threads, threads * sizeof(float)>>>(
        deviceBlueShadowArray,deviceBlueSumShadowArray_interm,imageSize);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());    
    threads = blocks; // launch one thread for each block in prev step
    blocks = 1;
    sum_up_arrays_by_reduction<<<blocks, threads, threads * sizeof(float)>>>(
    deviceBlueSumShadowArray_interm,deviceBlueSumShadowArray,imageSize); 
      CUDA_CHECK(hipGetLastError());
      CUDA_CHECK(hipDeviceSynchronize());

   // Launch sum_up_arrays kernel on the light arrays for each channel
    threads = maxThreadsPerBlock;
    blocks = imageSize / maxThreadsPerBlock;
    sum_up_arrays_by_reduction<<<blocks, threads, threads * sizeof(float)>>>(
        deviceRedLightArray,deviceRedSumLightArray_interm,imageSize);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());    
    threads = blocks; // launch one thread for each block in prev step
    blocks = 1;
    sum_up_arrays_by_reduction<<<blocks, threads, threads * sizeof(float)>>>(
    deviceRedSumLightArray_interm,deviceRedSumLightArray,imageSize); 
      CUDA_CHECK(hipGetLastError());
      CUDA_CHECK(hipDeviceSynchronize());
  
    threads = maxThreadsPerBlock;
    blocks = imageSize / maxThreadsPerBlock;
    sum_up_arrays_by_reduction<<<blocks, threads, threads * sizeof(float)>>>(
        deviceGreenLightArray,deviceGreenSumLightArray_interm,imageSize);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());    
    threads = blocks; // launch one thread for each block in prev step
    blocks = 1;
    sum_up_arrays_by_reduction<<<blocks, threads, threads * sizeof(float)>>>(
    deviceGreenSumLightArray_interm,deviceGreenSumLightArray,imageSize); 
      CUDA_CHECK(hipGetLastError());
      CUDA_CHECK(hipDeviceSynchronize());

    threads = maxThreadsPerBlock;
    blocks = imageSize / maxThreadsPerBlock;
    sum_up_arrays_by_reduction<<<blocks, threads, threads * sizeof(float)>>>(
        deviceBlueLightArray,deviceBlueSumLightArray_interm,imageSize);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());    
    threads = blocks; // launch one thread for each block in prev step
    blocks = 1;
    sum_up_arrays_by_reduction<<<blocks, threads, threads * sizeof(float)>>>(
    deviceBlueSumLightArray_interm,deviceBlueSumLightArray,imageSize); 
      CUDA_CHECK(hipGetLastError());
      CUDA_CHECK(hipDeviceSynchronize());

  // Launch sum_up_arrays kernel on the eroded shadow array
    threads = maxThreadsPerBlock;
    blocks = imageSize / maxThreadsPerBlock;
    sum_up_arrays_by_reduction1<<<blocks, threads, threads * sizeof(unsigned char)>>>(
        deviceErodedShadowMask,deviceErodedSumShadowArray_interm,imageSize);
      CUDA_CHECK(hipGetLastError());
      CUDA_CHECK(hipDeviceSynchronize());
    threads = blocks; // launch one thread for each block in prev step
    blocks = 1;
    sum_up_arrays_by_reduction<<<blocks, threads, threads * sizeof(float)>>>(
    deviceErodedSumShadowArray_interm,deviceErodedSumShadowArray,imageSize); 
      CUDA_CHECK(hipGetLastError());
      CUDA_CHECK(hipDeviceSynchronize());
  
  // Launch sum_up_arrays kernel on the eroded light array
    threads = maxThreadsPerBlock;
    blocks = imageSize / maxThreadsPerBlock;
    sum_up_arrays_by_reduction1<<<blocks, threads, threads * sizeof(unsigned char)>>>(
      deviceErodedLightMask,deviceErodedSumLightArray_interm,imageSize);
      CUDA_CHECK(hipGetLastError());
      CUDA_CHECK(hipDeviceSynchronize());  
    threads = blocks; // launch one thread for each block in prev step
    blocks = 1;
    sum_up_arrays_by_reduction<<<blocks, threads, threads * sizeof(float)>>>(
    deviceErodedSumLightArray_interm,deviceErodedSumLightArray,imageSize); 
      CUDA_CHECK(hipGetLastError());
      CUDA_CHECK(hipDeviceSynchronize());

  redSumShadowArray = (float *)malloc(imageSize * sizeof(float));
  greenSumShadowArray = (float *)malloc(imageSize * sizeof(float));
  blueSumShadowArray = (float *)malloc(imageSize * sizeof(float));
  redSumLightArray = (float *)malloc(imageSize * sizeof(float));
  greenSumLightArray = (float *)malloc(imageSize * sizeof(float));
  blueSumLightArray = (float *)malloc(imageSize * sizeof(float));
  erodedSumShadowArray = (float*)malloc(imageSize * sizeof(float));
  erodedSumLightArray = (float *)malloc(imageSize * sizeof(float));

  // Copy the GPU memory back to the CPU here
  wbTime_start(Copy, "Copying output memory to the CPU");
  CUDA_CHECK(hipMemcpy(redSumShadowArray, deviceRedSumShadowArray,
                        imageSize * sizeof(float),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(greenSumShadowArray, deviceGreenSumShadowArray,
                        imageSize * sizeof(float),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(blueSumShadowArray, deviceBlueSumShadowArray,
                        imageSize * sizeof(float),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(redSumLightArray, deviceRedSumLightArray,
                        imageSize * sizeof(float),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(greenSumLightArray, deviceGreenSumLightArray,
                        imageSize * sizeof(float),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(blueSumLightArray, deviceBlueSumLightArray,
                        imageSize * sizeof(float),
                        hipMemcpyDeviceToHost));  
  CUDA_CHECK(hipMemcpy(erodedSumShadowArray, deviceErodedSumShadowArray,
                        imageSize * sizeof(float),
                        hipMemcpyDeviceToHost));     
  CUDA_CHECK(hipMemcpy(erodedSumLightArray, deviceErodedSumLightArray,
                        imageSize * sizeof(float),
                        hipMemcpyDeviceToHost));                
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(Copy, "Copying output memory to the CPU");

  printf("\nSum of Red Shadow Array:\t");
  print_image(redSumShadowArray,imageWidth,imageHeight);
  printf("\nSum of Green Shadow Array:\t");
  print_image(greenSumShadowArray,imageWidth,imageHeight);
  printf("\nSum of Blue Shadow Array:\t");
  print_image(blueSumShadowArray,imageWidth,imageHeight);
  printf("\nSum of Red Light Array:\t");
  print_image(redSumLightArray,imageWidth,imageHeight);
  printf("\nSum of Green Light Array:\t");
  print_image(greenSumLightArray,imageWidth,imageHeight);
  printf("\nSum of Blue Light Array:\t");
  print_image(blueSumLightArray,imageWidth,imageHeight);
  printf("\nSum of Eroded  Shadow Array:\t");
  print_image(erodedSumShadowArray,imageWidth,imageHeight);
  printf("\nSum of Eroded  Light Array:\t");
  print_image(erodedSumLightArray,imageWidth,imageHeight);

  finalImage = (float *)malloc(imageSize * sizeof(float));

  wbTime_start(GPU, "Allocating GPU memory.");
  CUDA_CHECK( hipMalloc((void **)&deviceFinalImage, imageSize * sizeof(float)));  
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(GPU, "Allocating GPU memory.");

  // zero out bins
    CUDA_CHECK(hipMemset(deviceFinalImage, 0.0, imageSize * sizeof(float)));
  // Launch calculate_rgb_ratio kernel on the eroded shadow array and calculates the final image
  {
    dim3 blockDim(8,8), gridDim(1,1);
    calculate_final_image<<<gridDim, blockDim>>>(
    deviceRedSumShadowArray, deviceGreenSumShadowArray,deviceBlueSumShadowArray,
    deviceRedSumLightArray, deviceGreenSumLightArray,deviceBlueSumLightArray,
    deviceErodedSumShadowArray,deviceErodedSumLightArray,
    deviceRgbImage, deviceSmoothMask, deviceFinalImage,
    imageWidth, imageHeight, NUM_CHANNELS);
      CUDA_CHECK(hipGetLastError());
      CUDA_CHECK(hipDeviceSynchronize());
  } 

  //@@ Copy the GPU memory back to the CPU here
  wbTime_start(Copy, "Copying output memory to the CPU");
  CUDA_CHECK(hipMemcpy(finalImage, deviceFinalImage,
                        imageSize * sizeof(float),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(Copy, "Copying output memory to the CPU");

  printf("\nFinal image (shadowless):\n");
  print_image(finalImage,imageWidth,imageHeight);

  //@@ Free the GPU memory here
  wbTime_start(GPU, "Freeing GPU Memory");

  CUDA_CHECK(hipFree(deviceRgbImage));
  CUDA_CHECK(hipFree(deviceRedShadowArray));
  CUDA_CHECK(hipFree(deviceGreenShadowArray));
  CUDA_CHECK(hipFree(deviceBlueShadowArray));
  CUDA_CHECK(hipFree(deviceRedLightArray));
  CUDA_CHECK(hipFree(deviceGreenLightArray));
  CUDA_CHECK(hipFree(deviceBlueLightArray));
  CUDA_CHECK(hipFree(deviceErodedShadowMask));
  CUDA_CHECK(hipFree(deviceErodedLightMask));
  CUDA_CHECK(hipFree(deviceRedSumShadowArray));
  CUDA_CHECK(hipFree(deviceRedSumShadowArray_interm));
  CUDA_CHECK(hipFree(deviceGreenSumShadowArray));
  CUDA_CHECK(hipFree(deviceGreenSumShadowArray_interm));
  CUDA_CHECK(hipFree(deviceBlueSumShadowArray));
  CUDA_CHECK(hipFree(deviceBlueSumShadowArray_interm));
  CUDA_CHECK(hipFree(deviceRedSumLightArray)); 
  CUDA_CHECK(hipFree(deviceRedSumLightArray_interm)); 
  CUDA_CHECK(hipFree(deviceGreenSumLightArray));
  CUDA_CHECK(hipFree(deviceGreenSumLightArray_interm));
  CUDA_CHECK(hipFree(deviceBlueSumLightArray));
  CUDA_CHECK(hipFree(deviceBlueSumLightArray_interm));
  CUDA_CHECK(hipFree(deviceErodedSumShadowArray));
  CUDA_CHECK(hipFree(deviceErodedSumShadowArray_interm));
  CUDA_CHECK(hipFree(deviceErodedSumLightArray));
  CUDA_CHECK(hipFree(deviceErodedSumLightArray_interm));
  CUDA_CHECK(hipFree(deviceSmoothMask));
  CUDA_CHECK(hipFree(deviceFinalImage));
  wbTime_stop(GPU, "Freeing GPU Memory");

  free(finalImage);
  free(redShadowArray);
  free(greenShadowArray);
  free(blueShadowArray);
  free(redLightArray);
  free(greenLightArray);
  free(blueLightArray);
  free(redSumShadowArray);
  free(greenSumShadowArray);
  free(blueSumShadowArray);
  free(redSumLightArray); 
  free(greenSumLightArray);
  free(blueSumLightArray);
  free(erodedSumShadowArray);
  free(erodedSumLightArray);
  free(smoothMask);
}

int main(int argc, char *argv[]) {
  
  wbArg_t args;
  int imageWidth;
  int imageHeight;
  int imageSize;

  char *inputImageFile;

  wbImage_t inputImage_RGB;

  float* inputImage_RGB_float;

  args = wbArg_read(argc, argv); // parse the input arguments

  inputImageFile = wbArg_getInputFile(args, 0);
  inputImage_RGB = wbImport(inputImageFile);

  imageWidth = wbImage_getWidth(inputImage_RGB);
  imageHeight = wbImage_getHeight(inputImage_RGB);

  printf("\nRunning Result Integration unit test on image of %dx%d\n",
           imageWidth, imageHeight, NUM_CHANNELS);

  unsigned char erodedShadow[16] = {1, 1, 1, 1,
                                    1, 1, 1, 1,
                                    1, 1, 1, 1,
                                    1, 1, 1, 1};
  unsigned char erodedLight[16] =  {1, 1, 1, 1,
                                    1, 1, 1, 1,
                                    1, 1, 1, 1,
                                    1, 1, 1, 1};
  float smoothMask[16] =  {0.4444,0.6667,0.6667,0.4444,
                           0.6667,1.0000,1.0000,0.6667,
                           0.6667,1.0000,1.0000,0.6667,
                           0.4444,0.6667,0.6667,0.444};

  inputImage_RGB_float  = wbImage_getData(inputImage_RGB);

  print_image(inputImage_RGB_float,imageWidth,imageHeight);

  unit_test(inputImage_RGB_float,erodedShadow,erodedLight,smoothMask,imageWidth, imageHeight);

  free(inputImage_RGB_float);
  wbImage_delete(inputImage_RGB);

  return 0;

}