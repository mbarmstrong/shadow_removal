#include "hip/hip_runtime.h"

#include <wb.h>
#include "kernel.cu"
#include "kernel_reduction.cu"
#include "../globals.h"

void unit_test( float *rgbImage,unsigned char *erodedShadowMask,unsigned char *erodedLightMask, float *smoothMask,int imageWidth, int imageHeight) {

    float redSumShadowArray;
    float greenSumShadowArray;
    float blueSumShadowArray;
    float redSumLightArray; 
    float greenSumLightArray;
    float blueSumLightArray;
    float erodedSumShadowArray;
    float erodedSumLightArray;
    float *deviceRgbImage;
    float *deviceRedShadowArray;
    float *deviceGreenShadowArray;
    float *deviceBlueShadowArray;
    float *deviceRedLightArray;
    float *deviceGreenLightArray;
    float *deviceBlueLightArray;
    float *deviceRedSumShadowArray;
    float *deviceGreenSumShadowArray;
    float *deviceBlueSumShadowArray;
    float *deviceRedSumLightArray; 
    float *deviceGreenSumLightArray;
    float *deviceBlueSumLightArray;
    float *deviceErodedSumShadowArray;
    float *deviceErodedSumLightArray;
    float *deviceRedRatio;
    float *deviceGreenRatio;
    float *deviceBlueRatio;
    unsigned char *deviceErodedShadowMask;
    unsigned char *deviceErodedLightMask;
    float *deviceSmoothMask;
    float *deviceFinalImage;

  float *finalImage;

  int imageSize = imageHeight * imageWidth;

  wbTime_start(GPU, "Allocating GPU memory.");
  CUDA_CHECK( hipMalloc((void **)&deviceRgbImage, imageSize * NUM_CHANNELS *sizeof(float)));
  CUDA_CHECK( hipMalloc((void **)&deviceSmoothMask, imageSize * sizeof(float)));
  CUDA_CHECK( hipMalloc((void **)&deviceErodedShadowMask, imageSize * sizeof(unsigned char)));
  CUDA_CHECK( hipMalloc((void **)&deviceErodedLightMask, imageSize * sizeof(unsigned char)));
  CUDA_CHECK( hipMalloc((void **)&deviceRedShadowArray, imageSize * sizeof(float)));
  CUDA_CHECK( hipMalloc((void **)&deviceGreenShadowArray, imageSize * sizeof(float)));
  CUDA_CHECK( hipMalloc((void **)&deviceBlueShadowArray, imageSize * sizeof(float)));
  CUDA_CHECK( hipMalloc((void **)&deviceRedLightArray, imageSize * sizeof(float)));
  CUDA_CHECK( hipMalloc((void **)&deviceGreenLightArray, imageSize * sizeof(float)));
  CUDA_CHECK( hipMalloc((void **)&deviceBlueLightArray, imageSize * sizeof(float)));     
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(GPU, "Allocating GPU memory.");

  // Copy memory to the GPU here
  wbTime_start(GPU, "Copying input memory to the GPU.");
  CUDA_CHECK(hipMemcpy(deviceRgbImage, rgbImage, imageSize * NUM_CHANNELS * sizeof(float),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(deviceSmoothMask, smoothMask, imageSize * sizeof(float),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(deviceErodedShadowMask, erodedShadowMask, imageSize * sizeof(unsigned char),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(deviceErodedLightMask, erodedLightMask, imageSize * sizeof(unsigned char),
                        hipMemcpyHostToDevice));    
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  printf("\nGray Shadow Mask:\t");
  print_image(erodedShadowMask,imageWidth,imageHeight);

  printf("\nGray Light Mask:\t");
  print_image(erodedLightMask,imageWidth,imageHeight);
  
  // Launch multiple_rgbImage_byMask kernel on the bins
  {
    dim3 blockDim(8,8), gridDim(1,1);
    multiply_rgbImage_byMask<<<gridDim, blockDim>>>(
      deviceRgbImage, deviceErodedShadowMask, 
      deviceErodedLightMask, deviceRedShadowArray,deviceGreenShadowArray,deviceBlueShadowArray,
      deviceRedLightArray,deviceGreenLightArray,deviceBlueLightArray,imageWidth,imageHeight, NUM_CHANNELS);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
  }

  // Launch sum_up_arrays kernel on the light and shadow arrays for each channel
  redSumShadowArray = gpu_sum_reduce(deviceRedShadowArray, imageSize);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  greenSumShadowArray = gpu_sum_reduce(deviceGreenShadowArray, imageSize);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

 // Launch sum_up_arrays kernel on the light and shadow arrays for each channel
  blueSumShadowArray = gpu_sum_reduce(deviceBlueShadowArray, imageSize);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());    

 // Launch sum_up_arrays kernel on the shadow arrays for each channel
  redSumLightArray = gpu_sum_reduce(deviceRedLightArray, imageSize);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());    

 // Launch sum_up_arrays kernel on the light arrays for each channel
  greenSumLightArray = gpu_sum_reduce(deviceGreenLightArray, imageSize);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());    

  blueSumLightArray = gpu_sum_reduce(deviceBlueLightArray, imageSize); 
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());      

// Launch sum_up_arrays kernel on the eroded shadow array
  erodedSumShadowArray = gpu_sum_reduce(deviceErodedShadowMask, imageSize);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

// Launch sum_up_arrays kernel on the eroded light array
erodedSumLightArray = gpu_sum_reduce(deviceErodedLightMask, imageSize); 
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());  

  printf("\nSum of Red Shadow Array:\t %.04f",redSumShadowArray);
  printf("\nSum of Green Shadow Array:\t%.04f",greenSumShadowArray);
  printf("\nSum of Blue Shadow Array:\t %.04f",blueSumShadowArray);
  printf("\nSum of Red Light Array:\t %.04f",redSumLightArray);
  printf("\nSum of Green Light Array:\t %.04f",greenSumLightArray);
  printf("\nSum of Blue Light Array:\t%.04f",blueSumShadowArray);
  printf("\nSum of Eroded  Shadow Array:\t%.04f",erodedSumShadowArray);
  printf("\nSum of Eroded  Light Array:\t%.04f",erodedSumLightArray);

  float redRatio = (((redSumLightArray/erodedSumLightArray)/(redSumShadowArray/erodedSumShadowArray)) -1);
  float greenRatio = (((greenSumLightArray/erodedSumLightArray)/(greenSumShadowArray/erodedSumShadowArray)) -1);
  float blueRatio = (((blueSumLightArray/erodedSumLightArray)/(blueSumShadowArray/erodedSumShadowArray)) -1);
  
  printf("\nredRatio:\t%.04f",redRatio);
  printf("\ngreenRatio:\t%.04f",greenRatio);
  printf("\nblueRatio:\t%.04f",blueRatio);


  wbTime_start(GPU, "Allocating GPU memory.");
  CUDA_CHECK( hipMalloc((void **)&deviceRedRatio, sizeof(float)));   
  CUDA_CHECK( hipMalloc((void **)&deviceGreenRatio, sizeof(float)));   
  CUDA_CHECK( hipMalloc((void **)&deviceBlueRatio, sizeof(float)));  
  // CUDA_CHECK( hipMalloc((void **)&deviceRedSumShadowArray, sizeof(float)));   
  // CUDA_CHECK( hipMalloc((void **)&deviceGreenSumShadowArray, sizeof(float)));   
  // CUDA_CHECK( hipMalloc((void **)&deviceBlueSumShadowArray, sizeof(float)));   
  // CUDA_CHECK( hipMalloc((void **)&deviceRedSumLightArray, sizeof(float)));   
  // CUDA_CHECK( hipMalloc((void **)&deviceGreenSumLightArray, sizeof(float)));
  // CUDA_CHECK( hipMalloc((void **)&deviceBlueSumLightArray, sizeof(float))); 
  // CUDA_CHECK( hipMalloc((void **)&deviceErodedSumLightArray, sizeof(float))); 
  // CUDA_CHECK( hipMalloc((void **)&deviceErodedSumShadowArray, sizeof(float)));  
  CUDA_CHECK(hipDeviceSynchronize());
  CUDA_CHECK(hipGetLastError()); 
  wbTime_stop(GPU, "Allocating GPU memory."); 

  // Copy the GPU memory back to the CPU here
wbTime_start(Copy, "Copying host memory to the GPU");
CUDA_CHECK(hipMemcpy(deviceRedRatio, &redRatio,
                      sizeof(float),
                      hipMemcpyHostToDevice));
CUDA_CHECK(hipMemcpy(deviceGreenRatio, &greenRatio,
                      sizeof(float),
                      hipMemcpyHostToDevice));
CUDA_CHECK(hipMemcpy(deviceBlueRatio, &blueRatio,
                      sizeof(float),
                      hipMemcpyHostToDevice));
// CUDA_CHECK(hipMemcpy(deviceRedSumShadowArray, &redSumShadowArray,
//                       sizeof(float),
//                       hipMemcpyHostToDevice));
// CUDA_CHECK(hipMemcpy(deviceGreenSumShadowArray, &greenSumShadowArray,
//                       sizeof(float),
//                       hipMemcpyHostToDevice));
// CUDA_CHECK(hipMemcpy(deviceBlueSumShadowArray, &blueSumShadowArray,
//                       sizeof(float),
//                       hipMemcpyHostToDevice));
// CUDA_CHECK(hipMemcpy(deviceRedSumLightArray, &redSumLightArray,
//                       sizeof(float),
//                       hipMemcpyHostToDevice));
// CUDA_CHECK(hipMemcpy(deviceGreenSumLightArray, &greenSumLightArray,
//                       sizeof(float),
//                       hipMemcpyHostToDevice));
// CUDA_CHECK(hipMemcpy(deviceBlueSumLightArray, &blueSumLightArray,
//                       sizeof(float),
//                       hipMemcpyHostToDevice)); 
// CUDA_CHECK(hipMemcpy(deviceErodedSumShadowArray, &erodedSumShadowArray,
//                       sizeof(float),
//                       hipMemcpyHostToDevice));     
// CUDA_CHECK(hipMemcpy(deviceErodedSumLightArray, &erodedSumLightArray,
//                       sizeof(float),
//                       hipMemcpyHostToDevice));               
CUDA_CHECK(hipDeviceSynchronize());
wbTime_stop(Copy, "Copying output memory to the CPU");

  finalImage = (float *)malloc(imageSize * NUM_CHANNELS * sizeof(float));

  wbTime_start(GPU, "Allocating GPU memory.");
  CUDA_CHECK( hipMalloc((void **)&deviceFinalImage, imageSize * NUM_CHANNELS * sizeof(float)));  
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(GPU, "Allocating GPU memory.");

  // zero out bins
    CUDA_CHECK(hipMemset(deviceFinalImage, 0.0, imageSize * NUM_CHANNELS * sizeof(float)));
  // Launch calculate_rgb_ratio kernel on the eroded shadow array and calculates the final image
  {
  dim3 gridDim2(ceil(imageWidth/16.0), ceil(imageHeight/16.0), 1);
  dim3 blockDim2(16, 16, 1);
  calculate_final_image_stride<<<gridDim2, blockDim2>>>(deviceRedRatio, deviceGreenRatio,deviceBlueRatio,
  deviceRgbImage, deviceSmoothMask, deviceFinalImage,
  imageWidth, imageHeight, NUM_CHANNELS);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
  } 

  //@@ Copy the GPU memory back to the CPU here
  wbTime_start(Copy, "Copying output memory to the CPU");
  CUDA_CHECK(hipMemcpy(finalImage, deviceFinalImage,
                        imageSize * NUM_CHANNELS * sizeof(float),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(Copy, "Copying output memory to the CPU");

  printf("\nFinal image (shadowless):\n");
  print_image(finalImage,imageWidth,imageHeight);

  //@@ Free the GPU memory here
  wbTime_start(GPU, "Freeing GPU Memory");

  CUDA_CHECK(hipFree(deviceRgbImage));
  CUDA_CHECK(hipFree(deviceRedShadowArray));
  CUDA_CHECK(hipFree(deviceGreenShadowArray));
  CUDA_CHECK(hipFree(deviceBlueShadowArray));
  CUDA_CHECK(hipFree(deviceRedLightArray));
  CUDA_CHECK(hipFree(deviceGreenLightArray));
  CUDA_CHECK(hipFree(deviceBlueLightArray));
  CUDA_CHECK(hipFree(deviceErodedShadowMask));
  CUDA_CHECK(hipFree(deviceErodedLightMask));
  CUDA_CHECK(hipFree(deviceSmoothMask));
  CUDA_CHECK(hipFree(deviceFinalImage));
  wbTime_stop(GPU, "Freeing GPU Memory");

}

int main(int argc, char *argv[]) {
  
  wbArg_t args;
  int imageWidth;
  int imageHeight;

  char *inputImageFile;

  wbImage_t inputImage_RGB;

  float* inputImage_RGB_float;

  args = wbArg_read(argc, argv); // parse the input arguments

  inputImageFile = wbArg_getInputFile(args, 0);
  inputImage_RGB = wbImport(inputImageFile);

  imageWidth = wbImage_getWidth(inputImage_RGB);
  imageHeight = wbImage_getHeight(inputImage_RGB);

  printf("\nRunning Result Integration unit test on image of %dx%d\n",
           imageWidth, imageHeight, NUM_CHANNELS);

  unsigned char erodedShadow[16] = {1, 1, 1, 1,
                                    1, 1, 1, 1,
                                    1, 1, 1, 1,
                                    1, 1, 1, 1};
  unsigned char erodedLight[16] =  {1, 1, 1, 1,
                                    1, 1, 1, 1,
                                    1, 1, 1, 1,
                                    1, 1, 1, 1};
  float smoothMask[16] =  {0.4444,0.6667,0.6667,0.4444,
                           0.6667,1.0000,1.0000,0.6667,
                           0.6667,1.0000,1.0000,0.6667,
                           0.4444,0.6667,0.6667,0.444};

  inputImage_RGB_float  = wbImage_getData(inputImage_RGB);

  print_image(inputImage_RGB_float,imageWidth,imageHeight);

  unit_test(inputImage_RGB_float,erodedShadow,erodedLight,smoothMask,imageWidth, imageHeight);

  wbImage_delete(inputImage_RGB);

  return 0;

}