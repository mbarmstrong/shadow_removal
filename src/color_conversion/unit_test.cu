#include "hip/hip_runtime.h"
#include <wb.h>
#include "../globals.h"
#include "kernel.cu"

int main(int argc, char *argv[]) {
  
  	wbArg_t args;
  	int imageChannels;
  	int imageWidth;
  	int imageHeight;

  	char *inputImageFile;

	wbImage_t inputImage_RGB;
	wbImage_t outputImage_Inv;
	wbImage_t outputImage_Gray;
  	wbImage_t outputImage_YUV;

  	float *hostInputImageData_RGB;
  	float *hostOutputImageData_Inv;
  	float *hostOutputImageData_Gray;
  	float *hostOutputImageData_YUV;

  	float *deviceInputImageData_RGB;
  	float *deviceOutputImageData_Inv;
  	float *deviceOutputImageData_Gray;
  	float *deviceOutputImageData_YUV;

  	args = wbArg_read(argc, argv); // parse the input arguments

  	inputImageFile = wbArg_getInputFile(args, 0);
  	inputImage_RGB = wbImport(inputImageFile);

  	imageWidth = wbImage_getWidth(inputImage_RGB);
  	imageHeight = wbImage_getHeight(inputImage_RGB);
  	imageChannels = wbImage_getChannels(inputImage_RGB);

  	outputImage_Inv = wbImage_new(imageWidth, imageHeight, 3);
  	outputImage_Gray = wbImage_new(imageWidth, imageHeight, 3);
  	outputImage_YUV = wbImage_new(imageWidth, imageHeight, 3);

  	hostInputImageData_RGB = wbImage_getData(inputImage_RGB);
  	hostOutputImageData_Inv = wbImage_getData(outputImage_Inv);
  	hostOutputImageData_Gray = wbImage_getData(outputImage_Gray);
  	hostOutputImageData_YUV = wbImage_getData(outputImage_YUV);

  	wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

  	wbTime_start(GPU, "Doing GPU memory allocation");
  	hipMalloc((void **)&deviceInputImageData_RGB,
            	imageWidth * imageHeight * imageChannels * sizeof(float));
  	hipMalloc((void **)&deviceOutputImageData_Gray,
            	imageWidth * imageHeight * sizeof(float));
  	hipMalloc((void **)&deviceOutputImageData_YUV,
            	imageWidth * imageHeight * sizeof(float));
  	wbTime_stop(GPU, "Doing GPU memory allocation");

  	wbTime_start(Copy, "Copying data to the GPU");
  	hipMemcpy(deviceInputImageData_RGB, hostInputImageData_RGB,
            	imageWidth * imageHeight * imageChannels * sizeof(float),
            	hipMemcpyHostToDevice);
  	wbTime_stop(Copy, "Copying data to the GPU");

  	wbTime_start(Compute, "Doing the computation on the GPU");

  	// defining grid size (num blocks) and block size (num threads per block)
  	dim3 myGrid(ceil(imageWidth/16.0), ceil(imageHeight/16.0), 1);
  	dim3 myBlock(16, 16, 1);

  	// launch kernel
  	//color_convert<<<myGrid, myBlock>>>(deviceInputImageData_RGB, deviceOutputImageData_Inv, 
  	//								   deviceOutputImageData_Gray, deviceOutputImageData_YUV, 
  	//								   imageWidth, imageHeight, imageChannels);
	// convert_rgb_invariant<<<myGrid, myBlock>>>(deviceInputImageData_RGB, deviceOutputImageData_Inv, imageWidth, imageHeight, imageChannels);
	// convert_invariant_grayscale<<<myGrid, myBlock>>>(deviceOutputImageData_Inv, deviceOutputImageData_Gray, imageWidth, imageHeight, imageChannels);
 	// convert_rgb_yuv<<<myGrid, myBlock>>>(deviceInputImageData_RGB, deviceOutputImageData_YUV, imageWidth, imageHeight, imageChannels);
  	
  	wbTime_stop(Compute, "Doing the computation on the GPU");

  	wbTime_start(Copy, "Copying data from the GPU");
  	hipMemcpy(hostOutputImageData_Inv, deviceOutputImageData_Inv,
    		       imageWidth * imageHeight * sizeof(float), hipMemcpyDeviceToHost);
  	hipMemcpy(hostOutputImageData_Gray, deviceOutputImageData_Gray,
    		       imageWidth * imageHeight * sizeof(float), hipMemcpyDeviceToHost);
  	hipMemcpy(hostOutputImageData_YUV, deviceOutputImageData_YUV,
    		       imageWidth * imageHeight * sizeof(float), hipMemcpyDeviceToHost);
  	wbTime_stop(Copy, "Copying data from the GPU");

  	wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  	//wbSolution(args, outputImage_Inv, outputImage_Gray, outputImage_YUV);

  	hipFree(deviceInputImageData_RGB);
  	hipFree(deviceOutputImageData_Inv);
  	hipFree(deviceOutputImageData_Gray);
  	hipFree(deviceOutputImageData_YUV);

  	wbImage_delete(outputImage_Inv);
  	wbImage_delete(outputImage_Gray);
  	wbImage_delete(outputImage_YUV);
  	wbImage_delete(inputImage_RGB);

  	return 0;
}
