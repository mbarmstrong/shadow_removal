#include "hip/hip_runtime.h"
#include <wb.h>
#include "../globals.h"
#include "kernel.cu"

int main(int argc, char *argv[]) {
  
  	wbArg_t args;
  	int imageWidth;
  	int imageHeight;
    int imageSize;

  	char *inputImageFile;

	wbImage_t inputImage_RGB;
	wbImage_t outputImage_Inv;
	wbImage_t outputImage_Gray;
    wbImage_t outputImage_YUV;

  	float *hostInputImageData_RGB;
  	float *hostOutputImageData_Inv;
  	float *hostOutputImageData_Gray;
  	float *hostOutputImageData_YUV;

  	float *deviceInputImageData_RGB;
  	float *deviceOutputImageData_Inv;
  	float *deviceOutputImageData_Gray;
  	float *deviceOutputImageData_YUV;

  	args = wbArg_read(argc, argv); // parse the input arguments

    // FIXME: generate input image
  	inputImageFile = wbArg_getInputFile(args, 0);
  	inputImage_RGB = wbImport(inputImageFile);

  	imageWidth = wbImage_getWidth(inputImage_RGB);
  	imageHeight = wbImage_getHeight(inputImage_RGB);

    imageSize = imageWidth * imageHeight;

  	outputImage_Inv = wbImage_new(imageWidth, imageHeight, NUM_CHANNELS);
  	outputImage_Gray = wbImage_new(imageWidth, imageHeight, 1);
  	outputImage_YUV = wbImage_new(imageWidth, imageHeight, NUM_CHANNELS);

  	hostInputImageData_RGB = wbImage_getData(inputImage_RGB);

  	wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

    //@@ Allocate GPU memory here
  	wbTime_start(GPU, "Doing GPU memory allocation");
  	CUDA_CHECK(hipMalloc((void **)&deviceInputImageData_RGB, imageSize * NUM_CHANNELS * sizeof(float)));
  	CUDA_CHECK(hipMalloc((void **)&deviceOutputImageData_Gray, imageSize * 1 * sizeof(float)));
  	CUDA_CHECK(hipMalloc((void **)&deviceOutputImageData_YUV, imageSize * NUM_CHANNELS * sizeof(float)));
  	wbTime_stop(GPU, "Doing GPU memory allocation");

    //@@ Copy memory to the GPU here
  	wbTime_start(Copy, "Copying data to the GPU");
  	CUDA_CHECK(hipMemcpy(deviceInputImageData_RGB, hostInputImageData_RGB,
            	imageSize * NUM_CHANNELS * sizeof(float), hipMemcpyHostToDevice));
  	wbTime_stop(Copy, "Copying data to the GPU");

  	wbTime_start(Compute, "Doing the computation on the GPU");
  	// defining grid size (num blocks) and block size (num threads per block)
  	dim3 gridDim(ceil(imageWidth/16.0), ceil(imageHeight/16.0), 1);
  	dim3 blockDim(16, 16, 1);

  	// launch kernel
  	// color_convert<<<gridDim, blockDim>>>(deviceInputImageData_RGB, deviceOutputImageData_Inv, 
  	//								       deviceOutputImageData_Gray, deviceOutputImageData_YUV, 
  	//								       imageWidth, imageHeight, imageChannels);
  	wbTime_stop(Compute, "Doing the computation on the GPU");

    //@@ Copy the GPU memory back to the CPU here
  	wbTime_start(Copy, "Copying data from the GPU");
  	CUDA_CHECK(hipMemcpy(hostOutputImageData_Inv, deviceOutputImageData_Inv,
    		       imageSize * NUM_CHANNELS * sizeof(float), hipMemcpyDeviceToHost));
  	CUDA_CHECK(hipMemcpy(hostOutputImageData_Gray, deviceOutputImageData_Gray,
    		       imageSize * 1 * sizeof(float), hipMemcpyDeviceToHost));
  	CUDA_CHECK(hipMemcpy(hostOutputImageData_YUV, deviceOutputImageData_YUV,
    		       imageSize * NUM_CHANNELS * sizeof(float), hipMemcpyDeviceToHost));
  	wbTime_stop(Copy, "Copying data from the GPU");

  	wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  	// wbSolution(args, outputImage_Inv, outputImage_Gray, outputImage_YUV);

    //@@ Free the GPU memory here
    wbTime_start(GPU, "Freeing GPU Memory");
  	CUDA_CHECK(hipFree(deviceInputImageData_RGB));
  	CUDA_CHECK(hipFree(deviceOutputImageData_Inv));
  	CUDA_CHECK(hipFree(deviceOutputImageData_Gray));
  	CUDA_CHECK(hipFree(deviceOutputImageData_YUV));
    wbTime_stop(GPU, "Freeing GPU Memory");

  	wbImage_delete(outputImage_Inv);
  	wbImage_delete(outputImage_Gray);
  	wbImage_delete(outputImage_YUV);
  	wbImage_delete(inputImage_RGB);

  	return 0;
}
