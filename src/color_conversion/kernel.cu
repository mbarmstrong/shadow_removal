#include "hip/hip_runtime.h"
#include "../globals.h"

// rather than having three kernels to do each step individually, 
// merge them into	a single kernel -- this way we avoid reading input image
// multiple times by each kernel and increase flops per memory read

__global__ void color_convert(float *rgbImage, float *invImage, unsigned char *grayImage, unsigned char *yuvImage, int width, int height) {
	int col = threadIdx.x + blockIdx.x * blockDim.x; // column index
	int row = threadIdx.y + blockIdx.y * blockDim.y; // row index

    // we want to write back pixels so each channel stores all elements sequentially in mem.
	// this allows us to easily access a single channel for future processing.
	int stride = width*height;

	if (col < width && row < height) {	// check boundary condition
		int idx = row * width + col;   	// mapping 2D to 1D coordinate

		// load input RGB values
		float r = rgbImage[NUM_CHANNELS * idx];      // red component
        float g = rgbImage[NUM_CHANNELS * idx + 1];  // green component
        float b = rgbImage[NUM_CHANNELS * idx + 2];  // blue component

        // calculate RGB to color invariant
        float c1 = atan(r / max(g,b));
        float c2 = atan(g / max(r,b));
        float c3 = atan(b / max(r,g));

        // store new values in output invariant image
        invImage[idx]              = c1;	// FIXME: check indices
	    invImage[1 * stride + idx] = c2;
	    invImage[2 * stride + idx] = c3;

	    // calculate invariant to grayscale
	    // based off matlab function rgb2gray
	    // store new value in output grayscale image
        grayImage[idx] = round(((0.299 * c1) + (0.587 * c2) + (0.114 * c3)) * 255); 

	    // calculate RGB to YUV

	  	// based off matlab function rgb2ycbcr
	    unsigned char y = round((r * 65.481)  + (g * 128.553) + (b * 24.966)  + 16.0);  // luminance component
	    unsigned char u = round((r * -37.797) + (g * -74.203) + (b * 112.000) + 128.0);	// blue chrominance component
	    unsigned char v = round((r * 112.000) + (g * -93.786) + (b * -18.214) + 128.0);	// red chrominance component

	    //// based off nvidia function RGBToYCbCr
	    // float y = (r * 0.257)	+ (g * 0.504) 	+ (b * 0.098)	+ 16.0;  // luminance component
	    // float u = (r * -0.148) 	+ (g * -0.291)	+ (b * 0.439) 	+ 128.0; // blue chrominance component
	    // float v = (r * 0.439) 	+ (g * -0.368) 	+ (b * -0.071) 	+ 128.0; // red chrominance component

	    // store new values in output YUV image
	    yuvImage[idx]              = y;	// FIXME: check indices
	    yuvImage[1 * stride + idx] = u;
	    yuvImage[2 * stride + idx] = v; 

	}
}

// individual kernel -- RGB to invariant
// not used
__global__ void convert_rgb_invariant( float *rgbImage, float *invImage, int width, int height, int num_channels) {
  
  	// invariant: a feature that remains unchanged when a particular transformation is applied
	// "Color based object recognition," T. Gevers

    int col = threadIdx.x + blockIdx.x * blockDim.x; // column index
    int row = threadIdx.y + blockIdx.y * blockDim.y; // row index

    if (col < width && row < height) {  // check boundary condition
        int idx = row * width + col;   	// mapping 2D to 1D coordinate

        float r = rgbImage[num_channels * idx];      // red component
        float g = rgbImage[num_channels * idx + 1];  // green component
        float b = rgbImage[num_channels * idx + 2];  // blue component

        float c1 = atan(r / max(g,b));
        float c2 = atan(g / max(r,b));
        float c3 = atan(b / max(r,g));

        invImage[num_channels * idx]     = c1;
	    invImage[num_channels * idx + 1] = c2;
	    invImage[num_channels * idx + 2] = c3; 
    }
}

// individual kernel -- invariant to grayscale
// not used
__global__ void convert_invariant_grayscale(float *invImage, unsigned char *grayImage, int width, int height, int num_channels) {
  
    int col = threadIdx.x + blockIdx.x * blockDim.x; // column index
    int row = threadIdx.y + blockIdx.y * blockDim.y; // row index

    if (col < width && row < height) {	// check boundary condition
        int idx = row * width + col;  	// mapping 2D to 1D coordinate

        float r = invImage[num_channels * idx];      // red component
        float g = invImage[num_channels * idx + 1];  // green component
        float b = invImage[num_channels * idx + 2];  // blue component

        // rescale pixel using rgb values and floating point constants
        // store new pixel value in grayscale image
        grayImage[idx] = (0.21 * r) + (0.71 * g) + (0.07 * b); 
    }
}

// individual kernel -- RGB to YUV
// not used
__global__ void convert_rgb_yuv(float *rgbImage, unsigned char *yuvImage, int width, int height, int num_channels) {

  	int col = threadIdx.x + blockIdx.x * blockDim.x; // column index
  	int row = threadIdx.y + blockIdx.y * blockDim.y; // row index

  	if (col < width && row < height) {	// check boundary condition
	    int idx = row * width + col;   	// mapping 2D to 1D coordinate

	    // FIXME -- don't need to multiply by num channels since both have 3 channels?
	    float r = rgbImage[num_channels * idx];      // red component
	    float g = rgbImage[num_channels * idx + 1];  // green component
	    float b = rgbImage[num_channels * idx + 2];  // blue component

	    // Y range = [16,235], Cb range = Cr range = [16,240]
	    // Y values are conventionally shifted and scaled to the range [16, 235]
	    // rather than using the full range of [0, 255].
		// U and V values, which may be positive or negative, are summed with 128 
		// to make them always positive.

	  	// based off matlab function rgb2ycbcr
	    float y = (r * 65.481) 	+ (g * 128.553)	+ (b * 24.966) 	+ 16.0;		// luminance component
	    float u = (r * -37.797)	+ (g * -74.203)	+ (b * 112.000)	+ 128.0;	// blue chrominance component
	    float v = (r * 112.000)	+ (g * -93.786)	+ (b * -18.214)	+ 128.0;	// red chrominance component

	    // based off nvidia function RGBToYCbCr
	    y = (r * 0.257)	+ (g * 0.504) 	+ (b * 0.098)	+ 16.0;  	// luminance component
	    u = (r * -0.148) 	+ (g * -0.291)	+ (b * 0.439) 	+ 128.0;  	// blue chrominance component
	    v = (r * 0.439) 	+ (g * -0.368) 	+ (b * -0.071) 	+ 128.0;  	// red chrominance component

	    yuvImage[num_channels * idx]     = y;
	    yuvImage[num_channels * idx + 1] = u;
	    yuvImage[num_channels * idx + 2] = v; 
  	}
}