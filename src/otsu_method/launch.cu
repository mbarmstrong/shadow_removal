#include "hip/hip_runtime.h"

#include "kernel.cu"
#include "histo.cu"
#include "histo_thrust.cu"
#include "unit_test.cu"

#define RUN_SWEEPS_HISTO 0

float launch_otsu_method(unsigned char* image, int imageWidth, int imageHeight, const char* imageid) {

  unsigned int* deviceBins;
  unsigned char* deviceImage;

  float* deviceOmega;
  float* deviceMu;
  float* deviceSigmaBsq;
  float* hostSigmaBsq;

  int imageSize = imageWidth * imageHeight;

  hostSigmaBsq = (float *)malloc(NUM_BINS * sizeof(float));

  //@@ Allocate GPU memory here
  CUDA_CHECK( hipMalloc((void **)&deviceImage, imageSize * sizeof(unsigned char)) );
  CUDA_CHECK( hipMalloc((void **)&deviceBins, NUM_BINS * sizeof(unsigned int)) );
  CUDA_CHECK( hipMalloc((void **)&deviceOmega, NUM_BINS * sizeof(float)) );
  CUDA_CHECK( hipMalloc((void **)&deviceMu, NUM_BINS * sizeof(float)) );
  CUDA_CHECK( hipMalloc((void **)&deviceSigmaBsq, NUM_BINS * sizeof(float)) );
  CUDA_CHECK( hipDeviceSynchronize() );

  //@@ Copy memory to the GPU here
  CUDA_CHECK(hipMemcpy(deviceImage, image,
                        imageSize * sizeof(unsigned char),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipDeviceSynchronize());

  #if RUN_SWEEPS_HISTO
  printf("\nRunning Histogram Sweeps\n\n");
  histograms(deviceImage, deviceBins, imageWidth, imageHeight, imageid);
  histo_thrust(image, imageWidth, imageHeight, imageid);
  #endif

  // zero out bins
  CUDA_CHECK(hipMemset(deviceBins, 0, NUM_BINS * sizeof(unsigned int)));
  CUDA_CHECK(hipMemset(deviceOmega, 0.0, NUM_BINS * sizeof(float)));
  CUDA_CHECK(hipMemset(deviceMu, 0.0, NUM_BINS * sizeof(float)));
  CUDA_CHECK(hipMemset(deviceSigmaBsq, 0.0, NUM_BINS * sizeof(float)));

  // Launch histogram kernel on the bins
  dim3 blockDim(512), gridDim(30);
  timerLog_startEvent(&timerLog);
  histogram<<<gridDim, blockDim, NUM_BINS * sizeof(unsigned int)>>>(
      deviceImage, deviceBins, imageSize);
  timerLog_stopEventAndLog(&timerLog, "histogram", imageid, imageWidth, imageHeight);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());


  dim3 blockDim1(NUM_BINS), gridDim1(1);
  timerLog_startEvent(&timerLog);
  omega<<<gridDim1,blockDim1>>>(deviceBins,deviceOmega,imageSize);

  mu<<<gridDim1,blockDim1>>>(deviceBins,deviceMu,imageSize);

  sigma_b_squared<<<gridDim1,blockDim1>>>(deviceOmega,deviceMu,deviceSigmaBsq);

  //@@ Copy the GPU memory back to the CPU here
  CUDA_CHECK(hipMemcpy(hostSigmaBsq, deviceSigmaBsq,
                        NUM_BINS * sizeof(float),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipDeviceSynchronize());

  //Replace with kernel but probs won't gain much speedup
  float level = calculate_threshold_cpu(hostSigmaBsq);
  timerLog_stopEventAndLog(&timerLog, "threshold calculation", imageid, imageWidth, imageHeight);

  //@@ Free the GPU memory here
  wbTime_start(GPU, "Freeing GPU Memory");
  CUDA_CHECK(hipFree(deviceImage));
  CUDA_CHECK(hipFree(deviceBins));
  CUDA_CHECK(hipFree(deviceOmega));
  CUDA_CHECK(hipFree(deviceMu));
  CUDA_CHECK(hipFree(deviceSigmaBsq));
  wbTime_stop(GPU, "Freeing GPU Memory");

  free(hostSigmaBsq);

  return level;
}

void launch_image_binarization(unsigned char* image, unsigned char* binaryImage, float level, int imageWidth, int imageHeight, int flipped, const char* imageid) {

  unsigned char *deviceImage;
  unsigned char *deviceBinaryImage;

  int imageSize = imageWidth*imageHeight;
  
  CUDA_CHECK( hipMalloc((void **)&deviceImage, imageSize * sizeof(unsigned char)) );
  CUDA_CHECK( hipMalloc((void **)&deviceBinaryImage, imageSize * sizeof(float)) );
  CUDA_CHECK( hipDeviceSynchronize() );

  //@@ Copy memory to the GPU here
  CUDA_CHECK(hipMemcpy(deviceImage, image,
                        imageSize * sizeof(unsigned char),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipDeviceSynchronize());

  // zero out image
  CUDA_CHECK(hipMemset(deviceBinaryImage, 0, imageSize * sizeof(unsigned char)));

  int n_threads = 16;
  dim3 gridDim2(ceil((float)imageWidth/(float)n_threads),ceil((float)imageHeight/(float)n_threads));
  dim3 blockDim2(n_threads,n_threads);

  timerLog_startEvent(&timerLog);
  create_binarized_image<<<gridDim2, blockDim2>>>(deviceImage, deviceBinaryImage,
                                                level, imageWidth, imageHeight, flipped);
  timerLog_stopEventAndLog(&timerLog, "image binarization", imageid, imageWidth, imageHeight);

  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  //@@ Copy the GPU memory back to the CPU here
  CUDA_CHECK(hipMemcpy(binaryImage, deviceBinaryImage,
                        imageSize * sizeof(unsigned char),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipDeviceSynchronize());

  CUDA_CHECK(hipFree(deviceImage));
  CUDA_CHECK(hipFree(deviceBinaryImage));


}