#include "hip/hip_runtime.h"

#include <wb.h>
#include "kernel.cu"
#include "../globals.h"


float unit_test(unsigned char* image, int imageWidth, int imageHeight) {

  unsigned int* hostBins;
  unsigned int* deviceBins;
  unsigned char* deviceImage;

  int imageSize = imageWidth * imageHeight;

  hostBins = (unsigned int *)malloc(NUM_BINS * sizeof(unsigned int));

  //@@ Allocate GPU memory here
  wbTime_start(GPU, "Allocating GPU memory.");
  CUDA_CHECK( hipMalloc((void **)&deviceImage, imageSize * sizeof(unsigned char)) );
  CUDA_CHECK( hipMalloc((void **)&deviceBins, NUM_BINS * sizeof(unsigned int)) );
  CUDA_CHECK( hipDeviceSynchronize() );
  wbTime_stop(GPU, "Allocating GPU memory.");

  //@@ Copy memory to the GPU here
  wbTime_start(GPU, "Copying input memory to the GPU.");
  CUDA_CHECK(hipMemcpy(deviceImage, image,
                        imageSize * sizeof(unsigned char),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  // zero out bins
  CUDA_CHECK(hipMemset(deviceBins, 0, NUM_BINS * sizeof(unsigned int)));
  
  // Launch histogram kernel on the bins
  dim3 blockDim(512), gridDim(30);
  histogram<<<gridDim, blockDim, NUM_BINS * sizeof(unsigned int)>>>(
      deviceImage, deviceBins, imageSize);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  //@@ Copy the GPU memory back to the CPU here
  wbTime_start(Copy, "Copying output memory to the CPU");
  CUDA_CHECK(hipMemcpy(hostBins, deviceBins,
                        NUM_BINS * sizeof(unsigned int),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(Copy, "Copying output memory to the CPU");

  printf("histogram bins:");
  print_sparse_array(hostBins,NUM_BINS);

  float* deviceOmega;
  float* hostOmega;
  
  hostOmega = (float *)malloc(NUM_BINS * sizeof(float));

  wbTime_start(GPU, "Allocating GPU memory.");
  CUDA_CHECK( hipMalloc((void **)&deviceOmega, NUM_BINS * sizeof(float)) );
  CUDA_CHECK( hipDeviceSynchronize() );
  wbTime_stop(GPU, "Allocating GPU memory.");

  CUDA_CHECK(hipMemset(deviceOmega, 0.0, NUM_BINS * sizeof(float)));

  dim3 blockDim1(NUM_BINS), gridDim1(1);
  omega<<<gridDim1,blockDim1>>>(deviceBins,deviceOmega,imageSize);

  //@@ Copy the GPU memory back to the CPU here
  wbTime_start(Copy, "Copying output memory to the CPU");
  CUDA_CHECK(hipMemcpy(hostOmega, deviceOmega,
                        NUM_BINS * sizeof(float),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(Copy, "Copying output memory to the CPU");

  printf("\nomega:");
  print_step_array(hostOmega,NUM_BINS);

  float* deviceMu;
  float* hostMu;
  
  hostMu = (float *)malloc(NUM_BINS * sizeof(float));

  wbTime_start(GPU, "Allocating GPU memory.");
  CUDA_CHECK( hipMalloc((void **)&deviceMu, NUM_BINS * sizeof(float)) );
  CUDA_CHECK( hipDeviceSynchronize() );
  wbTime_stop(GPU, "Allocating GPU memory.");

  CUDA_CHECK(hipMemset(deviceMu, 0.0, NUM_BINS * sizeof(float)));

  mu<<<gridDim1,blockDim1>>>(deviceBins,deviceMu,imageSize);

  //@@ Copy the GPU memory back to the CPU here
  wbTime_start(Copy, "Copying output memory to the CPU");
  CUDA_CHECK(hipMemcpy(hostMu, deviceMu,
                        NUM_BINS * sizeof(float),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(Copy, "Copying output memory to the CPU");

  printf("\nmu:");
  print_step_array(hostMu,NUM_BINS);

  float* deviceSigmaBsq;
  float* hostSigmaBsq;
  
  hostSigmaBsq = (float *)malloc(NUM_BINS * sizeof(float));

  wbTime_start(GPU, "Allocating GPU memory.");
  CUDA_CHECK( hipMalloc((void **)&deviceSigmaBsq, NUM_BINS * sizeof(float)) );
  CUDA_CHECK( hipDeviceSynchronize() );
  wbTime_stop(GPU, "Allocating GPU memory.");

  CUDA_CHECK(hipMemset(deviceSigmaBsq, 0.0, NUM_BINS * sizeof(float)));

  sigma_b_squared<<<gridDim1,blockDim1>>>(deviceOmega,deviceMu,deviceSigmaBsq);

  //@@ Copy the GPU memory back to the CPU here
  wbTime_start(Copy, "Copying output memory to the CPU");
  CUDA_CHECK(hipMemcpy(hostSigmaBsq, deviceSigmaBsq,
                        NUM_BINS * sizeof(float),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(Copy, "Copying output memory to the CPU");

  printf("\nsigma_b_sq:");
  print_step_array(hostSigmaBsq,NUM_BINS);

  //Replace with kernel but probs won't gain much speedup
  float level = calculate_threshold_cpu(hostSigmaBsq);

  printf("\n level:\t%.4f\n\n", level);

  unsigned char *hostBinaryImage;
  unsigned char *deviceBinaryImage;

  hostBinaryImage = (unsigned char *)malloc(imageSize * sizeof(unsigned char));
  
  wbTime_start(GPU, "Allocating GPU memory.");
  CUDA_CHECK( hipMalloc((void **)&deviceBinaryImage, imageSize * sizeof(float)) );
  CUDA_CHECK( hipDeviceSynchronize() );
  wbTime_stop(GPU, "Allocating GPU memory.");

  // zero out image
  CUDA_CHECK(hipMemset(deviceBinaryImage, 0, imageSize * sizeof(unsigned char)));

  int n_threads = 16;
  dim3 gridDim2(ceil((float)imageWidth/(float)n_threads),ceil((float)imageHeight/(float)n_threads));
  dim3 blockDim2(n_threads,n_threads);
  create_binarized_image<<<gridDim2, blockDim2>>>(deviceImage, deviceBinaryImage,
                                                level, imageWidth, imageHeight, false);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  //@@ Copy the GPU memory back to the CPU here
  wbTime_start(Copy, "Copying output memory to the CPU");
  CUDA_CHECK(hipMemcpy(hostBinaryImage, deviceBinaryImage,
                        imageSize * sizeof(unsigned char),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(Copy, "Copying output memory to the CPU");

  print_image(hostBinaryImage,imageWidth,imageHeight);

  //@@ Free the GPU memory here
  wbTime_start(GPU, "Freeing GPU Memory");
  CUDA_CHECK(hipFree(deviceImage));
  CUDA_CHECK(hipFree(deviceBins));
  CUDA_CHECK(hipFree(deviceOmega));
  CUDA_CHECK(hipFree(deviceMu));
  CUDA_CHECK(hipFree(deviceSigmaBsq));
  CUDA_CHECK(hipFree(deviceBinaryImage));
  wbTime_stop(GPU, "Freeing GPU Memory");

  free(hostBins);
  free(hostOmega);
  free(hostMu);
  free(hostSigmaBsq);
  free(hostBinaryImage);

  return 0;

}


int main(int argc, char *argv[]) {
  
  	wbArg_t args;
  	int imageWidth;
  	int imageHeight;
    int imageSize;

  	char *inputImageFile;

  	wbImage_t inputImage_RGB;

    unsigned char* inputImage_RGB_uint8;

  	args = wbArg_read(argc, argv); // parse the input arguments

  	inputImageFile = wbArg_getInputFile(args, 0);
  	inputImage_RGB = wbImport(inputImageFile);

  	imageWidth = wbImage_getWidth(inputImage_RGB);
  	imageHeight = wbImage_getHeight(inputImage_RGB);

    imageSize = imageWidth * imageHeight;

    printf("\nRunning outsu unit test on image of %dx%d\n",
             imageWidth, imageHeight, NUM_CHANNELS);

    inputImage_RGB_uint8 = (unsigned char*)malloc(imageSize * sizeof(unsigned char));

    for(int i = 0; i < imageSize; i++){
        inputImage_RGB_uint8[i] = (unsigned char)(wbImage_getData(inputImage_RGB)[i*3]*255);
    }

    print_image(inputImage_RGB_uint8,imageWidth,imageHeight);

    unit_test(inputImage_RGB_uint8,imageWidth,imageHeight);

    free(inputImage_RGB_uint8);
    wbImage_delete(inputImage_RGB);

    return 0;

}