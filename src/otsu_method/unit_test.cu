#include "hip/hip_runtime.h"

#include <wb.h>
#include "kernel.cu"
#include "histo.cu"
#include "../globals.h"

st_timerLog_t timerLog;

void histograms(unsigned char* deviceImage, unsigned int* deviceBins, int imageSize) {

  hipEvent_t astartEvent, astopEvent;
  float aelapsedTime;
  hipEventCreate(&astartEvent);
  hipEventCreate(&astopEvent);
  
  // Launch histogram kernel on the bins
  dim3 blockDim(512), gridDim(30);

  CUDA_CHECK(hipMemset(deviceBins, 0, NUM_BINS * sizeof(unsigned int)));
  hipEventRecord(astartEvent, 0);
  wbTime_start(GPU, "Running global histogram");
  histogram_global_kernel<<<gridDim, blockDim, NUM_BINS * sizeof(unsigned int)>>>(
      deviceImage, deviceBins, imageSize);
  //CUDA_CHECK(hipGetLastError());
  //CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(GPU, "Running global histogram");
  hipEventRecord(astopEvent, 0);
  hipEventSynchronize(astopEvent);
  hipEventElapsedTime(&aelapsedTime, astartEvent, astopEvent);
  printf("\n");
  printf("Total compute time (ms) %f\n",aelapsedTime);
  printf("\n");
  timerLog_append(&timerLog,"histogram global",aelapsedTime);
  
  CUDA_CHECK(hipMemset(deviceBins, 0, NUM_BINS * sizeof(unsigned int)));
  hipEventRecord(astartEvent, 0);
  histogram_shared_kernel<<<gridDim, blockDim, NUM_BINS * sizeof(unsigned int)>>>(
      deviceImage, deviceBins, imageSize);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());
  hipEventRecord(astopEvent, 0);
  hipEventSynchronize(astopEvent);
  hipEventElapsedTime(&aelapsedTime, astartEvent, astopEvent);
  printf("\n");
  printf("Total compute time (ms) %f\n",aelapsedTime);
  printf("\n");
  timerLog_append(&timerLog,"histogram shared",aelapsedTime);

  CUDA_CHECK(hipMemset(deviceBins, 0, NUM_BINS * sizeof(unsigned int)));
  hipEventRecord(astartEvent, 0);
  histogram_shared_accumulate_kernel<<<gridDim, blockDim, NUM_BINS * sizeof(unsigned int)>>>(
      deviceImage, deviceBins, imageSize);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());
  hipEventRecord(astopEvent, 0);
  hipEventSynchronize(astopEvent);
  hipEventElapsedTime(&aelapsedTime, astartEvent, astopEvent);
  printf("\n");
  printf("Total compute time (ms) %f\n",aelapsedTime);
  printf("\n");
  timerLog_append(&timerLog,"histogram shared accumulate",aelapsedTime);

  CUDA_CHECK(hipMemset(deviceBins, 0, NUM_BINS * sizeof(unsigned int)));
  hipEventRecord(astartEvent, 0);
  wbTime_start(GPU, "Running R histogram");
  int shared_size = (NUM_BINS+1) * 12 * sizeof(unsigned int);
  histogram_shared_R_kernel<<<gridDim, blockDim, shared_size>>>(
        deviceImage, deviceBins, imageSize, 12);
  //CUDA_CHECK(hipGetLastError());
  //CUDA_CHECK(hipDeviceSynchronize());
  wbTime_stop(GPU, "Running R histogram");
  hipEventRecord(astopEvent, 0);
  hipEventSynchronize(astopEvent);
  hipEventElapsedTime(&aelapsedTime, astartEvent, astopEvent);
  printf("\n");
  printf("Total compute time (ms) %f\n",aelapsedTime);
  printf("\n");
  timerLog_append(&timerLog,"histogram shared R",aelapsedTime);

}

float unit_test(unsigned char* image, int imageWidth, int imageHeight) {


  //-------------------------------------------------
  //  Historgram
  //
  //-------------------------------------------------
  unsigned int* hostBins;
  unsigned int* deviceBins;
  unsigned char* deviceImage;

  int imageSize = imageWidth * imageHeight;

  hostBins = (unsigned int *)malloc(NUM_BINS * sizeof(unsigned int));

  //@@ Allocate GPU memory here
  CUDA_CHECK( hipMalloc((void **)&deviceImage, imageSize * sizeof(unsigned char)) );
  CUDA_CHECK( hipMalloc((void **)&deviceBins, NUM_BINS * sizeof(unsigned int)) );

  //@@ Copy memory to the GPU here
  CUDA_CHECK(hipMemcpy(deviceImage, image,
                        imageSize * sizeof(unsigned char),
                        hipMemcpyHostToDevice));

  histograms(deviceImage, deviceBins, imageSize);

  //@@ Copy the GPU memory back to the CPU here
  CUDA_CHECK(hipMemcpy(hostBins, deviceBins,
                        NUM_BINS * sizeof(unsigned int),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipDeviceSynchronize());

  printf("histogram bins:");
  print_sparse_array(hostBins,NUM_BINS);


  //-------------------------------------------------
  //  Omega
  //
  //-------------------------------------------------
  float* deviceOmega;
  float* hostOmega;
  
  hostOmega = (float *)malloc(NUM_BINS * sizeof(float));

  CUDA_CHECK( hipMalloc((void **)&deviceOmega, NUM_BINS * sizeof(float)) );
  CUDA_CHECK( hipDeviceSynchronize() );

  CUDA_CHECK(hipMemset(deviceOmega, 0.0, NUM_BINS * sizeof(float)));

  dim3 blockDim1(NUM_BINS), gridDim1(1);
  omega<<<gridDim1,blockDim1>>>(deviceBins,deviceOmega,imageSize);

  //@@ Copy the GPU memory back to the CPU here
  CUDA_CHECK(hipMemcpy(hostOmega, deviceOmega,
                        NUM_BINS * sizeof(float),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipDeviceSynchronize());

  printf("\nomega:");
  print_step_array(hostOmega,NUM_BINS);


  //-------------------------------------------------
  //  Mu
  //
  //-------------------------------------------------
  float* deviceMu;
  float* hostMu;
  
  hostMu = (float *)malloc(NUM_BINS * sizeof(float));

  CUDA_CHECK( hipMalloc((void **)&deviceMu, NUM_BINS * sizeof(float)) );
  CUDA_CHECK( hipDeviceSynchronize() );

  CUDA_CHECK(hipMemset(deviceMu, 0.0, NUM_BINS * sizeof(float)));

  mu<<<gridDim1,blockDim1>>>(deviceBins,deviceMu,imageSize);

  //@@ Copy the GPU memory back to the CPU here
  CUDA_CHECK(hipMemcpy(hostMu, deviceMu,
                        NUM_BINS * sizeof(float),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipDeviceSynchronize());

  printf("\nmu:");
  print_step_array(hostMu,NUM_BINS);


  //-------------------------------------------------
  //  Sigma B Squared
  //
  //-------------------------------------------------
  float* deviceSigmaBsq;
  float* hostSigmaBsq;
  
  hostSigmaBsq = (float *)malloc(NUM_BINS * sizeof(float));

  CUDA_CHECK( hipMalloc((void **)&deviceSigmaBsq, NUM_BINS * sizeof(float)) );
  CUDA_CHECK( hipDeviceSynchronize() );

  CUDA_CHECK(hipMemset(deviceSigmaBsq, 0.0, NUM_BINS * sizeof(float)));

  sigma_b_squared<<<gridDim1,blockDim1>>>(deviceOmega,deviceMu,deviceSigmaBsq);

  //@@ Copy the GPU memory back to the CPU here
  CUDA_CHECK(hipMemcpy(hostSigmaBsq, deviceSigmaBsq,
                        NUM_BINS * sizeof(float),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipDeviceSynchronize());

  printf("\nsigma_b_sq:");
  print_step_array(hostSigmaBsq,NUM_BINS);

  //-------------------------------------------------
  //  Calculate threashold level
  //
  //-------------------------------------------------

  //Replace with kernel but probs won't gain much speedup
  float level = calculate_threshold_cpu(hostSigmaBsq);

  printf("\n level:\t%.4f\n\n", level);


  //-------------------------------------------------
  //  Create binary image
  //
  //-------------------------------------------------
  unsigned char *hostBinaryImage;
  unsigned char *deviceBinaryImage;

  hostBinaryImage = (unsigned char *)malloc(imageSize * sizeof(unsigned char));
  
  CUDA_CHECK( hipMalloc((void **)&deviceBinaryImage, imageSize * sizeof(float)) );
  CUDA_CHECK( hipDeviceSynchronize() );

  // zero out image
  CUDA_CHECK(hipMemset(deviceBinaryImage, 0, imageSize * sizeof(unsigned char)));

  int n_threads = 16;
  dim3 gridDim2(ceil((float)imageWidth/(float)n_threads),ceil((float)imageHeight/(float)n_threads));
  dim3 blockDim2(n_threads,n_threads);
  create_binarized_image<<<gridDim2, blockDim2>>>(deviceImage, deviceBinaryImage,
                                                level, imageWidth, imageHeight, false);
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());

  //@@ Copy the GPU memory back to the CPU here
  CUDA_CHECK(hipMemcpy(hostBinaryImage, deviceBinaryImage,
                        imageSize * sizeof(unsigned char),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipDeviceSynchronize());

  //print_image(hostBinaryImage,imageWidth,imageHeight);

  //-------------------------------------------------
  //  Cleanup
  //
  //-------------------------------------------------
  //@@ Free the GPU memory here
  CUDA_CHECK(hipFree(deviceImage));
  CUDA_CHECK(hipFree(deviceBins));
  CUDA_CHECK(hipFree(deviceOmega));
  CUDA_CHECK(hipFree(deviceMu));
  CUDA_CHECK(hipFree(deviceSigmaBsq));
  CUDA_CHECK(hipFree(deviceBinaryImage));

  free(hostBins);
  free(hostOmega);
  free(hostMu);
  free(hostSigmaBsq);
  free(hostBinaryImage);

  return 0;

}


int main(int argc, char *argv[]) {
  
  	wbArg_t args;
  	int imageWidth;
  	int imageHeight;
    int imageSize;

  	char *inputImageFile;

  	wbImage_t inputImage_RGB;

    unsigned char* inputImage_RGB_uint8;

  	args = wbArg_read(argc, argv); // parse the input arguments

    timerLog = timerLog_new( wbArg_getOutputFile(args) );

  	inputImageFile = wbArg_getInputFile(args, 0);
  	inputImage_RGB = wbImport(inputImageFile);

  	imageWidth = wbImage_getWidth(inputImage_RGB);
  	imageHeight = wbImage_getHeight(inputImage_RGB);

    imageSize = imageWidth * imageHeight;

    printf("\nRunning outsu unit test on image of %dx%d\n",
             imageWidth, imageHeight, NUM_CHANNELS);

    inputImage_RGB_uint8 = (unsigned char*)malloc(imageSize * sizeof(unsigned char));

    for(int i = 0; i < imageSize; i++){
        inputImage_RGB_uint8[i] = (unsigned char)(wbImage_getData(inputImage_RGB)[i*3]*255);
    }

    //print_image(inputImage_RGB_uint8,imageWidth,imageHeight);

    unit_test(inputImage_RGB_uint8,imageWidth,imageHeight);

    timerLog_save(&timerLog);

    free(inputImage_RGB_uint8);
    wbImage_delete(inputImage_RGB);

    return 0;

}